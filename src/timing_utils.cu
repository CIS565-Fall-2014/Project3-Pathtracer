
// CUDA Dependencies
#include <hip/hip_runtime.h>

// Project Dependencies
#include "timing_utils.h"

//Global data
hipEvent_t beginEvent, endEvent;

void startTiming() {
	//Add timing options
	hipEventCreate( &beginEvent );
	hipEventCreate( &endEvent );

	//Execute the naive prefix sum and compute the time (in milliseconds)
	hipEventRecord(beginEvent, 0);
}

float stopTiming() {
	float time;

	hipEventRecord(endEvent, 0);
	hipEventSynchronize(endEvent);
	hipEventElapsedTime(&time, beginEvent, endEvent);

	//Cleanup timers
	hipEventDestroy(beginEvent);
	hipEventDestroy(endEvent);

	return time;
}