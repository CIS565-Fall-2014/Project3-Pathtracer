#include "hip/hip_runtime.h"
// CIS565 CUDA Raytracer: A parallel raytracer for Patrick Cozzi's CIS565: GPU Computing at the University of Pennsylvania
// Written by Yining Karl Li, Copyright (c) 2012 University of Pennsylvania
// This file includes code from:
//       Rob Farber for CUDA-GL interop, from CUDA Supercomputing For The Masses: http://www.drdobbs.com/architecture-and-design/cuda-supercomputing-for-the-masses-part/222600097
//       Peter Kutz and Yining Karl Li's GPU Pathtracer: http://gpupathtracer.blogspot.com/
//       Yining Karl Li's TAKUA Render, a massively parallel pathtracing renderer: http://www.yiningkarlli.com

#include <iostream>
#include <stdio.h>
#include <hip/hip_runtime.h>
#include <cmath>
#include "sceneStructs.h"
#include "glm/glm.hpp"
#include "utilities.h"
#include "raytraceKernel.h"
#include "intersections.h"
#include "interactions.h"

extern bool streamcompact_b;
extern bool texturemap_b;
extern bool bumpmap_b;
extern bool DOF_b;
extern bool MB_b;

void checkCUDAError(const char *msg) {
	hipError_t err = hipGetLastError();
	if( hipSuccess != err) {
		fprintf(stderr, "Cuda error: %s: %s.\n", msg, hipGetErrorString( err) ); 
		exit(EXIT_FAILURE); 
	}
} 

// LOOK: This function demonstrates how to use thrust for random number generation on the GPU!
// Function that generates static.
__host__ __device__ glm::vec3 generateRandomNumberFromThread(glm::vec2 resolution, float time, int x, int y){
	int index = x + (y * resolution.x);

	thrust::default_random_engine rng(hash(index*time));
	thrust::uniform_real_distribution<float> u01(0,1);

	return glm::vec3((float) u01(rng), (float) u01(rng), (float) u01(rng));
}

// TODO: IMPLEMENT THIS FUNCTION
// Function that does the initial raycast from the camera
__host__ __device__ ray raycastFromCameraKernel(glm::vec2 resolution, float time, int x, int y, glm::vec3 eye, glm::vec3 view, glm::vec3 up, glm::vec2 fov){  
	glm::vec3 A = glm::cross(view,up);
	glm::vec3 B = glm::cross(A,view);
	glm::vec3 M = eye + view;
	float angley = fov.y;
	float anglex= fov.x;
	glm::vec3 H =  glm::normalize(A) * glm::length(view) * tan(glm::radians(anglex));
	glm::vec3 V =  glm::normalize(B) * glm::length(view) * tan(glm::radians(angley));

	float sx = ((float)x)/(resolution.x - 1);
	float sy = ((float)y)/(resolution.y - 1);

	glm::vec3 P = M + (2.0f*sx-1)* H + (1-2.0f*sy) * V; //The picture begins
	glm::vec3 D = glm::normalize(P - eye);
	ray r;
	r.origin = eye;
	r.direction = D;
	return r;
}

//Kernel that blacks out a given image buffer
__global__ void clearImage(glm::vec2 resolution, glm::vec3* image){
	int x = (blockIdx.x * blockDim.x) + threadIdx.x;
	int y = (blockIdx.y * blockDim.y) + threadIdx.y;
	int index = x + (y * resolution.x);
	if(x<=resolution.x && y<=resolution.y){
		image[index] = glm::vec3(0,0,0);
	}
}

//Kernel that writes the image to the OpenGL PBO directly.
__global__ void sendImageToPBO(uchar4* PBOpos, glm::vec2 resolution, glm::vec3* image,float iterations){

	int x = (blockIdx.x * blockDim.x) + threadIdx.x;
	int y = (blockIdx.y * blockDim.y) + threadIdx.y;
	int index = x + (y * resolution.x);

	if(x<=resolution.x && y<=resolution.y){

		glm::vec3 color;
		color.x = image[index].x*255.0;
		color.y = image[index].y*255.0;
		color.z = image[index].z*255.0;
		color /= iterations;

		if(color.x>255){
			color.x = 255;
		}

		if(color.y>255){
			color.y = 255;
		}

		if(color.z>255){
			color.z = 255;
		}

		// Each thread writes one pixel location in the texture (textel)
		PBOpos[index].w = 0;
		PBOpos[index].x = color.x;
		PBOpos[index].y = color.y;
		PBOpos[index].z = color.z;
	}
}

// TODO: IMPLEMENT THIS FUNCTION
// Core raytracer kernel
__global__ void PathTraceColor(ray* remainrays,int raysnum,int currdepth,int maxdepth,
	staticGeom* geoms, int numberOfGeoms, int* lightIndex, 
	int lightNum,material* materials,float time,uint3* tcolors,int* tnums,bool textureb,uint3* bcolors,int* bnums,bool bumpb)
{
	int index = blockIdx.x * blockDim.x + threadIdx.x;
	if(index<=raysnum)
	{
		ray r = remainrays[index];
		if(!r.exist)  //If not exist, its new add color is 0
		{
			r.raycolor = glm::vec3(0,0,0);
			remainrays[index] = r;	
			return;
		}

		//clear all ray if currdepth == maxdepth
		if(currdepth==maxdepth)
		{
			r.exist = false;
			remainrays[index] = r;		
			return;
		}

		bool Intersect = false;
		glm::vec3 InterSectP,InterSectN;
		int IntersectgeomId = -1;
		Intersect = Intersecttest(r,InterSectP,InterSectN,geoms,numberOfGeoms,IntersectgeomId);
		if(bumpb)
			bumpMap(geoms,IntersectgeomId,InterSectN,InterSectP,bcolors,bnums);
		//if the ray intersect with nothing, give it black/backgroundcolor
		if(Intersect==false)
		{
			r.raycolor = glm::vec3(0,0,0);
			r.exist = false;
			remainrays[index] = r;		
			return;
		}


		material currMaterial = materials[geoms[IntersectgeomId].materialid];	
		if(textureb)
			textureMap(geoms,IntersectgeomId,currMaterial,InterSectN,InterSectP,tcolors,tnums);

		bool IsLight = false;
		for(int i=0;i<lightNum;i++)
		{
			if(IntersectgeomId==lightIndex[i])
				IsLight = true;
		}

		if(IsLight)
		{
			r.raycolor = r.raycolor * currMaterial.color * currMaterial.emittance;
			r.exist = false;
		}
		else
		{
			int seed = (index+1) * (time/2 + currdepth);
		    int BSDF = calculateBSDF(r,InterSectP,InterSectN,currMaterial,seed,currdepth);	
			r.raycolor = r.raycolor * currMaterial.color;
		}	

		remainrays[index] = r;		
	}
}

//Changed
__global__ void AddColor(glm::vec3* colors, ray* remainrays,int raysnum)
{
	int index = blockIdx.x * blockDim.x + threadIdx.x;

	if(index<=raysnum){
		ray r = remainrays[index];
		if(r.exist==false)
		   colors[r.initindex] += r.raycolor;
	}
}

__global__ void InitRays(ray* rays, glm::vec2 resolution, cameraData cam, float time,bool DOFbool)
{
	int x = (blockIdx.x * blockDim.x) + threadIdx.x;
	int y = (blockIdx.y * blockDim.y) + threadIdx.y;
	int index = x + (y * resolution.x);

	if((x<=resolution.x && y<=resolution.y))
	{
		//anti-aliasing
		thrust::default_random_engine rng(hash(index*time));
		thrust::uniform_real_distribution<float> u01(0, 1);
		ray r = raycastFromCameraKernel(resolution,0.0f, x + float(u01(rng)) -0.5f, y+float(u01(rng))-0.5f,cam.position,cam.view,cam.up,cam.fov);

		if(DOFbool)
	    {
		    glm::vec3 rand3 = generateRandomNumberFromThread(resolution, time, x, y);
		    glm::vec2 rand2 =  glm::vec2(rand3.x,rand3.y);
		    glm::vec3 offset = glm::vec3(rand2.x * cos((float)TWO_PI*rand2.y), rand2.x * sin((float)TWO_PI*rand2.y), 0.0f) * cam.blurradius;
		    glm::vec3 p = r.origin + r.direction * cam.focallength / glm::dot(cam.view, r.direction);
		    r.origin = r.origin + offset;
		    r.direction = glm::normalize(p - r.origin);
        }

		

		r.exist = true;
		r.initindex = index;
		r.raycolor = glm::vec3(1,1,1);
		r.IOR = 1.0f;
		rays[index] = r;
	}
}

struct Is_Exist
{
	__host__ __device__
	bool operator()(const ray x)
	{
		if(x.exist) return true;
		else return false;
	}
};

struct Is_Not_Exist
{
	__host__ __device__
	bool operator()(const ray x)
	{
		if(!x.exist) return true;
		else return false;
	}
};

//StreamC)ompact
void ThrustStreamCompact(thrust::device_ptr<ray> origin,int &N)
{
	//Count how many rays still exist
	int finallength = thrust::count_if(origin, origin+N,Is_Exist());
	thrust::remove_if(origin, origin+N,Is_Not_Exist());
	N = finallength;
	return;
}


// TODO: FINISH THIS FUNCTION
// Wrapper for the __global__ call that sets up the kernel calls and does a ton of memory management
void cudaRaytraceCore(uchar4* PBOpos, camera* renderCam, int frame, int iterations, material* materials,
	int numberOfMaterials, geom* geoms, int numberOfGeoms,std::vector<uint3> mapcolors,std::vector<int> maplastnums,
	std::vector<uint3> bumcolors,std::vector<int> bumlastnums){

	int traceDepth = 8; //determines how many bounces the raytracer traces

	// set up crucial magic
	int tileSize = 8;
	dim3 threadsPerBlock(tileSize, tileSize);
	dim3 fullBlocksPerGrid((int)ceil(float(renderCam->resolution.x)/float(tileSize)), (int)ceil(float(renderCam->resolution.y)/float(tileSize)));

	// send image to GPU
	glm::vec3* cudaimage = NULL;
	hipMalloc((void**)&cudaimage, (int)renderCam->resolution.x*(int)renderCam->resolution.y*sizeof(glm::vec3));
	hipMemcpy( cudaimage, renderCam->image, (int)renderCam->resolution.x*(int)renderCam->resolution.y*sizeof(glm::vec3), hipMemcpyHostToDevice);

	// package geometry and materials and sent to GPU
	staticGeom* geomList = new staticGeom[numberOfGeoms];
	for(int i=0; i<numberOfGeoms; i++){
		staticGeom newStaticGeom;
		newStaticGeom.type = geoms[i].type;
		newStaticGeom.materialid = geoms[i].materialid;
		newStaticGeom.translation = geoms[i].translations[frame];
		newStaticGeom.rotation = geoms[i].rotations[frame];
		newStaticGeom.scale = geoms[i].scales[frame];
		newStaticGeom.transform = geoms[i].transforms[frame];
		newStaticGeom.inverseTransform = geoms[i].inverseTransforms[frame];
		newStaticGeom.transinverseTransform = geoms[i].transinverseTransforms[frame];
		if(MB_b)
		{
			newStaticGeom.MBV = geoms[i].MBV[frame];
			int tempit = iterations%50;
			glm::mat4 transform;
			if(tempit>=0&&tempit<25)
			{
				newStaticGeom.translation += (float)tempit * newStaticGeom.MBV;
		        transform = utilityCore::buildTransformationMatrix(newStaticGeom.translation, newStaticGeom.rotation, newStaticGeom.scale);
			}
			else if(tempit>24)
			{
				newStaticGeom.translation -= (float)tempit * newStaticGeom.MBV;
		        transform = utilityCore::buildTransformationMatrix(newStaticGeom.translation, newStaticGeom.rotation, newStaticGeom.scale);
			}
			
		    newStaticGeom.transform = utilityCore::glmMat4ToCudaMat4(transform);
		    newStaticGeom.inverseTransform = utilityCore::glmMat4ToCudaMat4(glm::inverse(transform));
		}

		newStaticGeom.tri = geoms[i].tri;
		newStaticGeom.trinum = geoms[i].trinum;
		newStaticGeom.texindex = geoms[i].texindex;
		newStaticGeom.theight = geoms[i].theight;
		newStaticGeom.twidth = geoms[i].twidth;
		newStaticGeom.bumpindex = geoms[i].bumpindex;
		newStaticGeom.bheight = geoms[i].bheight;
		newStaticGeom.bwidth = geoms[i].bwidth;
		geomList[i] = newStaticGeom;
	}

	staticGeom* cudageoms = NULL;
	hipMalloc((void**)&cudageoms, numberOfGeoms*sizeof(staticGeom));
	hipMemcpy( cudageoms, geomList, numberOfGeoms*sizeof(staticGeom), hipMemcpyHostToDevice);

	//materials
	material* materialList = new material[numberOfMaterials];
	for(int i=0; i<numberOfMaterials; i++){
		material newMaterial;
		newMaterial.color = materials[i].color;

		//specular is useless as the highlight area color is decided by light
		newMaterial.specularExponent = materials[i].specularExponent;
		newMaterial.specularColor = materials[i].specularColor;  
		newMaterial.hasReflective = materials[i].hasReflective;
		newMaterial.hasRefractive = materials[i].hasRefractive;
		newMaterial.indexOfRefraction = materials[i].indexOfRefraction;
		newMaterial.hasScatter = materials[i].hasScatter;
		newMaterial.absorptionCoefficient = materials[i].absorptionCoefficient;
		newMaterial.reducedScatterCoefficient = materials[i].reducedScatterCoefficient;
		newMaterial.emittance = materials[i].emittance;
		materialList[i] = newMaterial;
	}

	material* cudamaterials = NULL;
	hipMalloc((void**)&cudamaterials, numberOfMaterials*sizeof(material));
	hipMemcpy( cudamaterials, materialList, numberOfMaterials*sizeof(material), hipMemcpyHostToDevice);

	//light
	int lcount = 0;
	for(int i=0; i<numberOfGeoms; i++)
	{
		if(materials[geomList[i].materialid].emittance>0)
			lcount++;
	}

	int *lightIds = new int[lcount];
	lcount = 0;
	for(int i=0; i<numberOfGeoms; i++)
	{
		if(materials[geomList[i].materialid].emittance>0)
		{
			lightIds[lcount] = i;
			lcount++;
		}
	}

	int *cudalightIds=NULL;
	hipMalloc((void**)&cudalightIds,lcount * sizeof(int));
	hipMemcpy( cudalightIds, lightIds,lcount * sizeof(int), hipMemcpyHostToDevice);

	// package camera
	cameraData cam;
	cam.resolution = renderCam->resolution;
	cam.position = renderCam->positions[frame];
	cam.view = renderCam->views[frame];
	cam.up = renderCam->ups[frame];
	cam.fov = renderCam->fov;
	cam.focallength = renderCam->focall;
	cam.blurradius = renderCam->blurr;

	//Transfer Texture Map
	uint3* cudacolors = NULL;
	int* cudalastnums = NULL;
	if(texturemap_b)
	{
		if(iterations==1  && (maplastnums.size()==0||mapcolors.size()==0))
		{
			std::cout<<"No Texture Map Set!"<<std::endl;
			texturemap_b = false;
		}
		uint3 *allcolors = new uint3[(int)mapcolors.size()];
	    int *alllastnum = new int[(int)maplastnums.size()];
        for(int i=0;i<(int)mapcolors.size();i++)
		    allcolors[i] = mapcolors[i];

	    for(int i=0;i<(int)maplastnums.size();i++)
		    alllastnum[i] = maplastnums[i];


	    hipMalloc((void**)&cudacolors, (int)mapcolors.size()*sizeof(uint3));
	    hipMemcpy( cudacolors, allcolors, (int)mapcolors.size()*sizeof(uint3), hipMemcpyHostToDevice);
		
	    hipMalloc((void**)&cudalastnums, (int)maplastnums.size()*sizeof(int));
	    hipMemcpy( cudalastnums, alllastnum, (int)maplastnums.size()*sizeof(int), hipMemcpyHostToDevice);
		delete allcolors;
		delete alllastnum;
	}
	
	//Transfer Bump Map
	uint3* cudabumcolors = NULL;
	int* cudabumlastnums = NULL;
	if(bumpmap_b)
	{
		if(iterations==1  && (bumlastnums.size()==0||bumcolors.size()==0))
		{
			std::cout<<"No Bump Map Set!"<<std::endl;
			bumpmap_b = false;
		}
		uint3 *allcolors = new uint3[(int)bumcolors.size()];
	    int *alllastnum = new int[(int)bumlastnums.size()];
        for(int i=0;i<(int)bumcolors.size();i++)
		    allcolors[i] = bumcolors[i];

	    for(int i=0;i<(int)bumlastnums.size();i++)
		    alllastnum[i] = bumlastnums[i];


	    hipMalloc((void**)&cudabumcolors, (int)bumcolors.size()*sizeof(uint3));
	    hipMemcpy( cudabumcolors, allcolors, (int)bumcolors.size()*sizeof(uint3), hipMemcpyHostToDevice);
		
	    hipMalloc((void**)&cudabumlastnums, (int)bumlastnums.size()*sizeof(int));
	    hipMemcpy( cudabumlastnums, alllastnum, (int)bumlastnums.size()*sizeof(int), hipMemcpyHostToDevice);
		delete allcolors;
		delete alllastnum;
	}

	//set up init rays
	int numberOfInitrays = renderCam->resolution.x*renderCam->resolution.y;
	ray* cudarays = NULL;
	hipMalloc((void**)&cudarays, numberOfInitrays*sizeof(ray));
	InitRays<<<fullBlocksPerGrid, threadsPerBlock>>>(cudarays,renderCam->resolution,cam,(float)iterations,DOF_b);

	//set path trace dim
	int raythreadsPerBlock = (int)(tileSize*tileSize);
	int rayblocksPerGrid = ceil((float)numberOfInitrays/(float)raythreadsPerBlock);

	// kernel launches
	for(int i=0;i<=traceDepth;i++)
	{
		if(numberOfInitrays>0)
		{
			PathTraceColor<<<rayblocksPerGrid, raythreadsPerBlock>>>(cudarays,numberOfInitrays,i,traceDepth,cudageoms,
				numberOfGeoms,cudalightIds,lcount,cudamaterials,(float)iterations,cudacolors,cudalastnums,texturemap_b
				,cudabumcolors,cudabumlastnums,bumpmap_b);
			AddColor<<<rayblocksPerGrid, raythreadsPerBlock>>>(cudaimage, cudarays,numberOfInitrays);
			if(streamcompact_b)
			{
				thrust::device_ptr<ray> rayStart(cudarays);
			    ThrustStreamCompact(rayStart,numberOfInitrays);
				rayblocksPerGrid = ceil((float)numberOfInitrays/(float)raythreadsPerBlock);
			}			
		}	
	}


	sendImageToPBO<<<fullBlocksPerGrid, threadsPerBlock>>>(PBOpos, renderCam->resolution, cudaimage,(float)iterations);

	// retrieve image from GPU
	hipMemcpy( renderCam->image, cudaimage, (int)renderCam->resolution.x*(int)renderCam->resolution.y*sizeof(glm::vec3), hipMemcpyDeviceToHost);

	// free up stuff, or else we'll leak memory like a madman
	hipFree( cudaimage );
	hipFree( cudageoms );
	//Added
	hipFree( cudalightIds );
	hipFree( cudamaterials );
	hipFree( cudarays );
	hipFree( cudacolors );
	hipFree( cudalastnums );
	hipFree( cudabumcolors );
	hipFree( cudabumlastnums );

	delete geomList;
	//Added
	delete materialList;
	delete lightIds;
	

	// make certain the kernel has completed
	hipDeviceSynchronize();

	checkCUDAError("Kernel failed!");
}