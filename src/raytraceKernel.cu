#include "hip/hip_runtime.h"
// CIS565 CUDA Raytracer: A parallel raytracer for Patrick Cozzi's CIS565: GPU Computing at the University of Pennsylvania
// Written by Yining Karl Li, Copyright (c) 2012 University of Pennsylvania
// This file includes code from:
//       Rob Farber for CUDA-GL interop, from CUDA Supercomputing For The Masses: http://www.drdobbs.com/architecture-and-design/cuda-supercomputing-for-the-masses-part/222600097
//       Peter Kutz and Yining Karl Li's GPU Pathtracer: http://gpupathtracer.blogspot.com/
//       Yining Karl Li's TAKUA Render, a massively parallel pathtracing renderer: http://www.yiningkarlli.com

#include <stdio.h>
#include <hip/hip_runtime.h>
#include <cmath>

#include "sceneStructs.h"
#include "glm/glm.hpp"
#include "utilities.h"
#include "raytraceKernel.h"
#include "intersections.h"
#include "interactions.h"


void checkCUDAError(const char *msg)
{
    hipError_t err = hipGetLastError();
    if ( hipSuccess != err) {
        fprintf(stderr, "Cuda error: %s: %s.\n", msg, hipGetErrorString( err) );
        exit(EXIT_FAILURE);
    }
}

// LOOK: This function demonstrates how to use thrust for random number generation on the GPU!
// Function that generates static.
__host__ __device__ glm::vec3 generateRandomNumberFromThread(glm::vec2 resolution, float time, int x, int y)
{
    int index = x + (y * resolution.x);

    thrust::default_random_engine rng(hash(index * time));
    thrust::uniform_real_distribution<float> u01(0, 1);

    return glm::vec3((float) u01(rng), (float) u01(rng), (float) u01(rng));
}

// Function that does the initial raycast from the camera
__host__ __device__ ray raycastFromCameraKernel(glm::vec2 resolution, float time, int x, int y, glm::vec3 eye, glm::vec3 view, glm::vec3 up, glm::vec2 fov)
{
    std::cout << fov.x << " " << fov.y << std::endl;
    fov *= PI / 180.f;
    glm::vec2 ndc = glm::vec2(1 - x / resolution.x * 2, 1 - y / resolution.y * 2);
    glm::vec3 dir = glm::normalize(view);
    glm::vec3 norX = glm::normalize(glm::cross(dir , up )) * glm::tan(fov.x);
    glm::vec3 norY = glm::normalize(glm::cross(norX, dir)) * glm::tan(fov.y);

#if 0
    // This is probably totally wrong but is mainly here for checking to make
    // sure that the time-averaging code results in convergence
    const float BLUR = 0.02f;
    thrust::default_random_engine rng(hash((x + y * resolution.x) * time));
    thrust::uniform_real_distribution<float> u(-BLUR, BLUR);
    thrust::uniform_real_distribution<float> v(-BLUR, BLUR);
    glm::vec3 lens = norX * u(rng) + norY * v(rng);
#else
    glm::vec3 lens;
#endif

    ray r;
    r.origin = eye + lens;
    r.direction = glm::normalize(dir + lens + norX * ndc.x + norY * ndc.y);
    return r;
}

//Kernel that blacks out a given image buffer
__global__ void clearImage(glm::vec2 resolution, glm::vec3* image)
{
    int x = (blockIdx.x * blockDim.x) + threadIdx.x;
    int y = (blockIdx.y * blockDim.y) + threadIdx.y;
    int index = x + (y * resolution.x);
    if (x <= resolution.x && y <= resolution.y) {
        image[index] = glm::vec3(0, 0, 0);
    }
}

//Kernel that writes the image to the OpenGL PBO directly.
__global__ void sendImageToPBO(uchar4* PBOpos, glm::vec2 resolution, glm::vec3* image)
{

    int x = (blockIdx.x * blockDim.x) + threadIdx.x;
    int y = (blockIdx.y * blockDim.y) + threadIdx.y;
    int index = x + (y * resolution.x);

    if (x <= resolution.x && y <= resolution.y) {

        glm::vec3 color;
        color.x = image[index].x * 255.0;
        color.y = image[index].y * 255.0;
        color.z = image[index].z * 255.0;

        if (color.x > 255) {
            color.x = 255;
        }

        if (color.y > 255) {
            color.y = 255;
        }

        if (color.z > 255) {
            color.z = 255;
        }

        // Each thread writes one pixel location in the texture (textel)
        PBOpos[index].w = 0;
        PBOpos[index].x = color.x;
        PBOpos[index].y = color.y;
        PBOpos[index].z = color.z;
    }
}


struct pathray {
    bool alive;
    int index;
    int depth;
    glm::vec3 color;
    ray r;
};

struct pathray_is_dead {
    __host__ __device__ bool operator()(const struct pathray pr) {
        return !pr.alive;
    }
};

__global__ void init_pathrays(struct pathray *pathrays, float time, cameraData cam)
{
    int index = (blockIdx.x * blockDim.x) + threadIdx.x;
    int x = index % (int) cam.resolution.x;
    int y = index / (int) cam.resolution.x;

    if (x < cam.resolution.x && y < cam.resolution.y) {
        struct pathray pr ={
            true, index, 0, glm::vec3(1, 1, 1),
            raycastFromCameraKernel(cam.resolution, time, x, y, cam.position, cam.view, cam.up, cam.fov),
        };
        pathrays[index] = pr;
    }
}

__device__ void merge_pathray(const struct pathray &pr, float time, glm::vec3 *colors)
{
    colors[pr.index] = (colors[pr.index] * time + pr.color) / (time + 1);
}

__global__ void merge_dead_pathrays(struct pathray *pathrays, int pathraycount, float time, glm::vec3 *colors)
{
    int index = (blockIdx.x * blockDim.x) + threadIdx.x;
    if (index >= pathraycount) {
        return;
    }

    struct pathray pr = pathrays[index];
    if (!pr.alive) {
        merge_pathray(pr, time, colors);
    }
    pathrays[index] = pr;
}

__global__ void merge_live_pathrays(struct pathray *pathrays, int pathraycount, float time, glm::vec3 *colors)
{
    int index = (blockIdx.x * blockDim.x) + threadIdx.x;
    if (index >= pathraycount) {
        return;
    }

    struct pathray pr = pathrays[index];
    if (pr.alive) {
        // If the path never hit a light, assume it's 0 for now.
        // TODO: take a direct path to the light sources?
        pr.color = glm::vec3();
        merge_pathray(pr, time, colors);
    }
}

__global__ void pathray_step(struct pathray *pathrays, int pathraycount, float time,
                            staticGeom* geoms, int numberOfGeoms,
                            staticGeom* lights, int numberOfLights,
                            material *mats, int numberOfMaterials)
{
    int index = (blockIdx.x * blockDim.x) + threadIdx.x;
    if (index >= pathraycount) {
        return;
    }

    struct pathray pr = pathrays[index];
    if (!pr.alive) {
        return;
    }

    ray r = pr.r;
    staticGeom tmin_geom;
    glm::vec3 tmin_pos;
    glm::vec3 tmin_nor;
    float tmin = 1e38f;
    for (int i = 0; i < numberOfGeoms; ++i) {
        glm::vec3 p;
        glm::vec3 n;
        float t = 2e38f;
        staticGeom g = geoms[i];
        if (g.type == SPHERE) {
            t = sphereIntersectionTest(g, r, p, n);
        } else if (g.type == CUBE) {
            t = boxIntersectionTest(g, r, p, n);
        } else if (g.type == MESH) {
        }
        if (t > 0 && t < tmin) {
            tmin = t;
            tmin_geom = g;
            tmin_pos = p;
            tmin_nor = n;
        }
    }

    if (tmin > 9e37) {
        // Empty space; abort ray
        pr.alive = false;
        // TODO: add some here instead of treating as black?
        pr.color = glm::vec3();
        pathrays[index] = pr;
        return;
    }

    material mat = mats[tmin_geom.materialid];

    if (mat.emittance) {
        // Hit a light; abort ray
        pr.alive = false;
        pr.color *= mat.emittance * mat.color;
        pathrays[index] = pr;
        return;
    }

    // Calculate the ray of the next bounce
    thrust::default_random_engine rng(hash(index * time));
    thrust::uniform_real_distribution<float> u01(0, 1);
    float branchcount = 2.f;
    float raytype = u01(rng) * branchcount;
    glm::vec3 c(branchcount);
    if (raytype < 1) {
        // Next bounce is diffuse
        c *= mat.color;
        pr.r.direction = calculateRandomDirectionInHemisphere(
                tmin_nor, u01(rng), u01(rng));
        pr.r.origin = tmin_pos + pr.r.direction * 0.001f;
    } else if (raytype < 2) {
        // Next bounce is specular... or reflective? Something.
        c *= mat.specularColor * mat.hasReflective;
        pr.r.direction = calculateReflectionDirection(tmin_nor, r.direction);
        pr.r.origin = tmin_pos + pr.r.direction * 0.001f;
    }
    pr.color *= c;
    pathrays[index] = pr;
}

// TODO: FINISH THIS FUNCTION
// Wrapper for the __global__ call that sets up the kernel calls and does a ton of memory management
void cudaRaytraceCore(uchar4* PBOpos, camera* renderCam, int frame, int iterations, material* materials, int numberOfMaterials, geom* geoms, int numberOfGeoms)
{
    const int traceDepth = 4; //determines how many bounces the raytracer traces
    const int pixelcount = ((int) renderCam->resolution.x) * ((int) renderCam->resolution.y);

    // set up crucial magic
    int tileSize = 8;
    dim3 threadsPerBlock(tileSize, tileSize);
    dim3 fullBlocksPerGrid((int)ceil(float(renderCam->resolution.x) / float(tileSize)), (int)ceil(float(renderCam->resolution.y) / float(tileSize)));

    // send image to GPU
    glm::vec3* cudaimage = NULL;
    hipMalloc((void**)&cudaimage, (int)renderCam->resolution.x * (int)renderCam->resolution.y * sizeof(glm::vec3));
    hipMemcpy( cudaimage, renderCam->image, (int)renderCam->resolution.x * (int)renderCam->resolution.y * sizeof(glm::vec3), hipMemcpyHostToDevice);

    // package geometry and materials and sent to GPU
    staticGeom* geomList = new staticGeom[numberOfGeoms];
    staticGeom* lightList = new staticGeom[numberOfGeoms];
    int numberOfLights = 0;
    for (int i = 0; i < numberOfGeoms; i++) {
        staticGeom newStaticGeom;
        newStaticGeom.type = geoms[i].type;
        newStaticGeom.materialid = geoms[i].materialid;
        newStaticGeom.translation = geoms[i].translations[frame];
        newStaticGeom.rotation = geoms[i].rotations[frame];
        newStaticGeom.scale = geoms[i].scales[frame];
        newStaticGeom.transform = geoms[i].transforms[frame];
        newStaticGeom.inverseTransform = geoms[i].inverseTransforms[frame];
        geomList[i] = newStaticGeom;
        if (materials[newStaticGeom.materialid].emittance > 0) {
            lightList[numberOfLights] = newStaticGeom;
            numberOfLights += 1;
        }
    }

    staticGeom* cudageoms = NULL;
    hipMalloc((void**)&cudageoms, numberOfGeoms * sizeof(staticGeom));
    hipMemcpy(cudageoms, geomList, numberOfGeoms * sizeof(staticGeom), hipMemcpyHostToDevice);
    staticGeom* cudalights = NULL;
    hipMalloc((void**)&cudalights, numberOfLights * sizeof(staticGeom));
    hipMemcpy(cudalights, lightList, numberOfLights * sizeof(staticGeom), hipMemcpyHostToDevice);

    material* cudamats = NULL;
    hipMalloc((void**)&cudamats, numberOfMaterials * sizeof(material));
    hipMemcpy(cudamats, materials, numberOfMaterials * sizeof(material), hipMemcpyHostToDevice);

    struct pathray *pathrays = NULL;
    hipMalloc((void **) &pathrays, pixelcount * sizeof(struct pathray));

    // package camera
    cameraData cam;
    cam.resolution = renderCam->resolution;
    cam.position = renderCam->positions[frame];
    cam.view = renderCam->views[frame];
    cam.up = renderCam->ups[frame];
    cam.fov = renderCam->fov;

    // kernel launches
    const int TPB = 256;
    int pathraycount = pixelcount;
    int bc = (pathraycount + TPB - 1) / TPB;
    float time = iterations;
    init_pathrays<<<bc, TPB>>>(pathrays, time, cam);
    for (int depth = 0; depth < traceDepth; ++depth) {
        // Compute one ray along each path
        pathray_step<<<bc, TPB>>>(pathrays, pathraycount, time,
            cudageoms, numberOfGeoms,
            cudalights, numberOfLights,
            cudamats, numberOfMaterials);
        // Merge all of the dead paths into the image
        merge_dead_pathrays<<<bc, TPB>>>(pathrays, pathraycount, time, cudaimage);
        // Stream compact all of the dead paths away
        // TODO: enable this later
        //pathraycount = thrust::remove_if(thrust::device,
        //        pathrays, pathrays + pathraycount, pathray_is_dead());
        bc = (pathraycount + TPB - 1) / TPB;
    }
    // And finally handle all of the paths that haven't died yet
    merge_live_pathrays<<<bc, TPB>>>(pathrays, pathraycount, time, cudaimage);

    sendImageToPBO<<<fullBlocksPerGrid, threadsPerBlock>>>(PBOpos, renderCam->resolution, cudaimage);

    // retrieve image from GPU
    hipMemcpy( renderCam->image, cudaimage, (int)renderCam->resolution.x * (int)renderCam->resolution.y * sizeof(glm::vec3), hipMemcpyDeviceToHost);

    // free up stuff, or else we'll leak memory like a madman
    hipFree(cudaimage);
    hipFree(cudageoms);
    hipFree(cudamats);
    hipFree(pathrays);
    delete[] geomList;

    // make certain the kernel has completed
    hipDeviceSynchronize();

    checkCUDAError("Kernel failed!");
}
