#include "hip/hip_runtime.h"
// CIS565 CUDA Raytracer: A parallel raytracer for Patrick Cozzi's CIS565: GPU Computing at the University of Pennsylvania
// Written by Yining Karl Li, Copyright (c) 2012 University of Pennsylvania
// This file includes code from:
//       Rob Farber for CUDA-GL interop, from CUDA Supercomputing For The Masses: http://www.drdobbs.com/architecture-and-design/cuda-supercomputing-for-the-masses-part/222600097
//       Peter Kutz and Yining Karl Li's GPU Pathtracer: http://gpupathtracer.blogspot.com/
//       Yining Karl Li's TAKUA Render, a massively parallel pathtracing renderer: http://www.yiningkarlli.com

#include <stdio.h>
#include <hip/hip_runtime.h>
#include <cmath>

#include "sceneStructs.h"
#include "glm/glm.hpp"
#include "utilities.h"
#include "raytraceKernel.h"
#include "intersections.h"
#include "interactions.h"

void checkCUDAError(const char *msg) {
  hipError_t err = hipGetLastError();
  if( hipSuccess != err) {
    fprintf(stderr, "Cuda error: %s: %s.\n", msg, hipGetErrorString( err) ); 
    exit(EXIT_FAILURE); 
  }
} 

// LOOK: This function demonstrates how to use thrust for random number generation on the GPU!
// Function that generates static.
__host__ __device__ glm::vec3 generateRandomNumberFromThread(glm::vec2 resolution, float time, int x, int y){
  int index = x + (y * resolution.x);
   
  thrust::default_random_engine rng(hash(index*time));
  thrust::uniform_real_distribution<float> u01(0,1);

  return glm::vec3((float) u01(rng), (float) u01(rng), (float) u01(rng));
}

// TODO: IMPLEMENT THIS FUNCTION
// Function that does the initial raycast from the camera
__host__ __device__ ray raycastFromCameraKernel(glm::vec2 resolution, float time, int x, int y, glm::vec3 eye, glm::vec3 view, glm::vec3 up, glm::vec2 fov){
  ray r;
  r.origin = eye;
  glm::vec3 image_x_direction=glm::cross(view,up);
  glm::vec3 image_y_direction=-up;
  glm::vec3 image_center=eye+view;
  float image_x=((float)x-(float)resolution.x/2)/((float)resolution.x/2);
  float image_y=((float)y-(float)resolution.y/2)/((float)resolution.y/2);
  float angle_x=fov.x;
  float angle_y=fov.y;
  glm::vec3 image_pos=image_center+image_x*glm::length(view)*tan(angle_x)*glm::normalize(image_x_direction)+image_y*glm::length(view)*tan(angle_y)*glm::normalize(image_y_direction);
  glm::vec3 ray_direction=glm::normalize(image_pos-eye);
  r.direction=ray_direction;
  return r;
}

//Kernel that blacks out a given image buffer
__global__ void clearImage(glm::vec2 resolution, glm::vec3* image){
    int x = (blockIdx.x * blockDim.x) + threadIdx.x;
    int y = (blockIdx.y * blockDim.y) + threadIdx.y;
    int index = x + (y * resolution.x);
    if(x<=resolution.x && y<=resolution.y){
      image[index] = glm::vec3(0,0,0);
    }
}

//Kernel that writes the image to the OpenGL PBO directly.
__global__ void sendImageToPBO(uchar4* PBOpos, glm::vec2 resolution, glm::vec3* image){
  
  int x = (blockIdx.x * blockDim.x) + threadIdx.x;
  int y = (blockIdx.y * blockDim.y) + threadIdx.y;
  int index = x + (y * resolution.x);
  
  if(x<=resolution.x && y<=resolution.y){

      glm::vec3 color;
      color.x = image[index].x*255.0;
      color.y = image[index].y*255.0;
      color.z = image[index].z*255.0;

      if(color.x>255){
        color.x = 255;
      }

      if(color.y>255){
        color.y = 255;
      }

      if(color.z>255){
        color.z = 255;
      }
      
      // Each thread writes one pixel location in the texture (textel)
      PBOpos[index].w = 0;
      PBOpos[index].x = color.x;
      PBOpos[index].y = color.y;
      PBOpos[index].z = color.z;
  }
}

// TODO: IMPLEMENT THIS FUNCTION
// Core raytracer kernel
__global__ void raytraceRay(glm::vec2 resolution, float time, cameraData cam, int rayDepth, glm::vec3* colors,
                            staticGeom* geoms, int numberOfGeoms){

  int x = (blockIdx.x * blockDim.x) + threadIdx.x;
  int y = (blockIdx.y * blockDim.y) + threadIdx.y;
  int index = x + (y * resolution.x);                                                                                       

  if((x<=resolution.x && y<=resolution.y)){
	//test for direction
	//ray newRay = raycastFromCameraKernel(resolution,time,x,y,cam.position,cam.view,cam.up,cam.fov);
	//colors[index]=255.0f*newRay.direction;
	colors[index]=glm::vec3(100.0f,100.0f,100.0f);
   }
}

// TODO: FINISH THIS FUNCTION
// Wrapper for the __global__ call that sets up the kernel calls and does a ton of memory management
void cudaRaytraceCore(uchar4* PBOpos, camera* renderCam, int frame, int iterations, material* materials, int numberOfMaterials, geom* geoms, int numberOfGeoms){
  // send image to GPU
  glm::vec3* cudaimage = NULL;
  hipMalloc((void**)&cudaimage, (int)renderCam->resolution.x*(int)renderCam->resolution.y*sizeof(glm::vec3));
  hipMemcpy( cudaimage, renderCam->image, (int)renderCam->resolution.x*(int)renderCam->resolution.y*sizeof(glm::vec3), hipMemcpyHostToDevice);
  
  // package geometry and materials and sent to GPU
  staticGeom* geomList = new staticGeom[numberOfGeoms];
  for(int i=0; i<numberOfGeoms; i++){
    staticGeom newStaticGeom;
    newStaticGeom.type = geoms[i].type;
    newStaticGeom.materialid = geoms[i].materialid;
    newStaticGeom.translation = geoms[i].translations[frame];
    newStaticGeom.rotation = geoms[i].rotations[frame];
    newStaticGeom.scale = geoms[i].scales[frame];
    newStaticGeom.transform = geoms[i].transforms[frame];
    newStaticGeom.inverseTransform = geoms[i].inverseTransforms[frame];
    geomList[i] = newStaticGeom;
  }
  //send geometry
  staticGeom* cudageoms = NULL;
  hipMalloc((void**)&cudageoms, numberOfGeoms*sizeof(staticGeom));
  hipMemcpy( cudageoms, geomList, numberOfGeoms*sizeof(staticGeom), hipMemcpyHostToDevice);
  //send materials
  material* cudamaterials=NULL;
  hipMalloc((void**)&cudamaterials,numberOfMaterials*sizeof(material));
  hipMemcpy(cudamaterials,materials,numberOfMaterials*sizeof(material),hipMemcpyHostToDevice);

  // package camera
  cameraData cam;
  cam.resolution = renderCam->resolution;
  cam.position = renderCam->positions[frame];
  cam.view = renderCam->views[frame];
  cam.up = renderCam->ups[frame];
  cam.fov = renderCam->fov;

  ///////////////////////////////////////////////////////////////////////////////////////////////////////////////////
  int traceDepth = 1; //determines how many bounces the raytracer traces
  // set up crucial magic
  int tileSize = 8;
  dim3 threadsPerBlock(tileSize, tileSize);
  dim3 fullBlocksPerGrid((int)ceil(float(renderCam->resolution.x)/float(tileSize)), (int)ceil(float(renderCam->resolution.y)/float(tileSize)));
  

  // kernel launches
  raytraceRay<<<fullBlocksPerGrid, threadsPerBlock>>>(renderCam->resolution, (float)iterations, cam, traceDepth, cudaimage, cudageoms, numberOfGeoms);

  sendImageToPBO<<<fullBlocksPerGrid, threadsPerBlock>>>(PBOpos, renderCam->resolution, cudaimage);

  // retrieve image from GPU
  hipMemcpy( renderCam->image, cudaimage, (int)renderCam->resolution.x*(int)renderCam->resolution.y*sizeof(glm::vec3), hipMemcpyDeviceToHost);

  // free up stuff, or else we'll leak memory like a madman
  hipFree( cudaimage );
  hipFree( cudageoms );
  delete geomList;

  // make certain the kernel has completed
  hipDeviceSynchronize();

  checkCUDAError("Kernel failed!");
}
