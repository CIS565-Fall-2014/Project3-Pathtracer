#include "hip/hip_runtime.h"
// CIS565 CUDA Raytracer: A parallel raytracer for Patrick Cozzi's CIS565: GPU Computing at the University of Pennsylvania
// Written by Yining Karl Li, Copyright (c) 2012 University of Pennsylvania
// This file includes code from:
//       Rob Farber for CUDA-GL interop, from CUDA Supercomputing For The Masses: http://www.drdobbs.com/architecture-and-design/cuda-supercomputing-for-the-masses-part/222600097
//       Peter Kutz and Yining Karl Li's GPU Pathtracer: http://gpupathtracer.blogspot.com/
//       Yining Karl Li's TAKUA Render, a massively parallel pathtracing renderer: http://www.yiningkarlli.com

#include <stdio.h>
#include <hip/hip_runtime.h>
#include <cmath>

#include <thrust/device_ptr.h>
#include <thrust/remove.h>

#include "sceneStructs.h"
#include "glm/glm.hpp"
#include "utilities.h"
#include "raytraceKernel.h"
#include "intersections.h"
#include "interactions.h"


// Some new types.
enum MaterialType {
	IDEAL_DIFFUSE,
	PERFECT_SPUCULAR,
	GLASS
};


// Some forward declarations.
__host__ __device__ bool sceneIntersection( const ray &r, staticGeom *geoms, int num_geoms, float &t, int &id, glm::vec3 &intersection_point, glm::vec3 &intersection_normal );
__host__ __device__ MaterialType determineMaterialType( material mat );
__host__ __device__ glm::vec2 computePixelSubsampleLocation( glm::vec2 pixel_index, glm::vec2 image_resolution, int current_iteration );


void checkCUDAError( const char *msg )
{
	hipError_t err = hipGetLastError();
	if ( hipSuccess != err) {
		fprintf( stderr, "Cuda error: %s: %s.\n", msg, hipGetErrorString( err ) ); 
		exit( EXIT_FAILURE );
	}
}


// LOOK: This function demonstrates how to use thrust for random number generation on the GPU!
// Function that generates static.
__host__
__device__
glm::vec3 generateRandomNumberFromThread( glm::vec2 resolution,
										  float time,
										  int x,
										  int y )
{
	int index = x + ( y * resolution.x );
   
	thrust::default_random_engine rng( hash( index * time ) );
	thrust::uniform_real_distribution<float> u01( 0, 1 );

	return glm::vec3( ( float )u01( rng ),
					  ( float )u01( rng ),
					  ( float )u01( rng ) );
}


// Kernel that blacks out a given image buffer
__global__
void clearImage( glm::vec2 resolution,
				 glm::vec3* image )
{
	int x = ( blockIdx.x * blockDim.x ) + threadIdx.x;
	int y = ( blockIdx.y * blockDim.y ) + threadIdx.y;
	int index = x + ( y * resolution.x );
	if ( x <= resolution.x && y <= resolution.y ) {
		image[index] = glm::vec3( 0, 0, 0 );
	}
}


// Kernel that writes the image to the OpenGL PBO directly.
__global__
void sendImageToPBO( uchar4* PBOpos,
					 glm::vec2 resolution,
					 glm::vec3* image,
					 int iterations_so_far )
{  
	int x = ( blockIdx.x * blockDim.x ) + threadIdx.x;
	int y = ( blockIdx.y * blockDim.y ) + threadIdx.y;
	int index = x + ( y * resolution.x );
  
	if ( x <= resolution.x && y <= resolution.y ) {
		glm::vec3 color;
		color.x = ( image[index].x / iterations_so_far ) * 255.0;
		color.y = ( image[index].y / iterations_so_far ) * 255.0;
		color.z = ( image[index].z / iterations_so_far ) * 255.0;

		if ( color.x > 255 ) {
			color.x = 255;
		}
		if ( color.y > 255 ) {
			color.y = 255;
		}
		if ( color.z > 255 ) {
			color.z = 255;
		}
      
		// Each thread writes one pixel location in the texture (textel)
		PBOpos[index].w = 0;
		PBOpos[index].x = color.x;
		PBOpos[index].y = color.y;
		PBOpos[index].z = color.z;
	}
}


/*********** CORE PATHTRACING ALGORITHMS ***********/


__host__
__device__
bool sceneIntersection( const ray &r,
						staticGeom *geoms,
						int num_geoms,
						float &t,
						int &id,
						glm::vec3 &intersection_point,
						glm::vec3 &intersection_normal )
{
	t = FLT_MAX;
	float temp_t = -1.0f;
	glm::vec3 temp_intersection_point;
	glm::vec3 temp_intersection_normal;

	// Find nearest intersection, if any.
	for ( int i = 0; i < num_geoms; ++i ) {
		if ( geoms[i].type == SPHERE ) {
			temp_t = sphereIntersectionTest( geoms[i],
											 r,
											 temp_intersection_point,
											 temp_intersection_normal );
		}
		else if ( geoms[i].type == CUBE ) {
			temp_t = boxIntersectionTest( geoms[i],
										  r,
										  temp_intersection_point,
										  temp_intersection_normal );
		}

		// Update nearest intersection if closer intersection has been found.
		if ( temp_t > 0.0f && temp_t < t ) {
			t = temp_t;
			intersection_point = temp_intersection_point;
			intersection_normal = temp_intersection_normal;
			id = geoms[i].materialid;
		}
	}

	return ( t < FLT_MAX );
}


//__host__
//__device__
//bool lightIntersection( const ray &r,
//						staticGeom *geoms,
//						int num_geoms,
//						int &obj_id )
//{
//	float t = FLT_MAX;
//	float temp_t = -1.0f;
//	glm::vec3 temp_intersection_point;
//	glm::vec3 temp_intersection_normal;
//
//	// Find nearest intersection, if any.
//	for ( int i = 0; i < num_geoms; ++i ) {
//		if ( geoms[i].type == SPHERE ) {
//			temp_t = sphereIntersectionTest( geoms[i],
//											 r,
//											 temp_intersection_point,
//											 temp_intersection_normal );
//		}
//		else if ( geoms[i].type == CUBE ) {
//			temp_t = boxIntersectionTest( geoms[i],
//										  r,
//										  temp_intersection_point,
//										  temp_intersection_normal );
//		}
//
//		// Update nearest intersection if closer intersection has been found.
//		if ( temp_t > 0.0f && temp_t < t ) {
//			t = temp_t;
//			obj_id = i;
//		}
//	}
//
//	return ( t < FLT_MAX );
//}


// Compute rays from camera through pixels and store in ray_pool.
__global__
void raycastFromCameraKernel( ray *ray_pool,
							 glm::vec2 resolution,
							 glm::vec3 eyep,
							 glm::vec3 m,
							 glm::vec3 h,
							 glm::vec3 v,
							 int current_iteration )
{
	int x = ( blockIdx.x * blockDim.x ) + threadIdx.x;
	int y = ( blockIdx.y * blockDim.y ) + threadIdx.y;
	int index = ( y * ( int )resolution.x ) + x;

	if ( index > ( resolution.x * resolution.y ) ) {
		return;
	}

	// TODO: Add a switch to turn anti-aliasing on/off.
	glm::vec2 pixel_location = computePixelSubsampleLocation( glm::vec2( x, y ), resolution, current_iteration );

	float sx = pixel_location.x / ( resolution.x - 1.0f );
	float sy = 1.0f - ( pixel_location.y / ( resolution.y - 1.0f ) );

	//float sx = ( float )x / ( resolution.x - 1.0f );
	//float sy = 1.0f - ( ( float )y / ( resolution.y - 1.0f ) );

	glm::vec3 image_point = m + ( ( 2.0f * sx - 1.0f ) * h ) + ( ( 2.0f * sy - 1.0f ) * v );

	glm::vec3 dir = image_point - eyep;

	ray r;
	r.origin = eyep;
	r.direction = glm::normalize( dir );
	r.image_coords = glm::vec2( x, y );

	ray_pool[index] = r;
}


__host__
__device__
glm::vec2 computePixelSubsampleLocation( glm::vec2 pixel_index,
										 glm::vec2 image_resolution,
										 int current_iteration )
{
	const int SUBSAMPLE_ROWS = 4;
	const int SUBSAMPLE_COLS = 4;

	float row_height = 1.0f / SUBSAMPLE_ROWS;
	float col_width = 1.0f / SUBSAMPLE_COLS;

	float left_bounds, upper_bounds;
	float x_percentage, y_percentage, new_x, new_y;

	int sub_pixel_num = current_iteration % ( SUBSAMPLE_ROWS * SUBSAMPLE_COLS );
	
	// partition pixel into grid using perfect_square and randomly sample one ray through each grid space
	upper_bounds = ( float )pixel_index.y;
	for ( int y = 0; y < SUBSAMPLE_ROWS; ++y ) {
		left_bounds = ( float )pixel_index.x;
		for ( int x = 0; x < SUBSAMPLE_COLS; ++x ) {
			if ( ( y * SUBSAMPLE_COLS ) + x == sub_pixel_num ) {
				glm::vec3 rand = generateRandomNumberFromThread( image_resolution, current_iteration, pixel_index.x, pixel_index.y );
				x_percentage = rand.x;
				y_percentage = rand.y;

				new_x = left_bounds + ( col_width * x_percentage );
				new_y = upper_bounds + ( row_height * y_percentage );

				return glm::vec2( new_x, new_y );
			}
			left_bounds += col_width;
		}
		upper_bounds += row_height;
	}
	return pixel_index;
}


//// Test kernel to verify raycastFromCameraKernel results were correct.
//__global__
//void testOutputKernel( glm::vec3 *image,
//					   ray *ray_pool,
//					   glm::vec2 resolution )
//{
//	int x = ( blockIdx.x * blockDim.x ) + threadIdx.x;
//	int y = ( blockIdx.y * blockDim.y ) + threadIdx.y;
//	int index = ( y * ( int )resolution.x ) + x;
//
//	if ( index > ( resolution.x * resolution.y ) ) {
//		return;
//	}
//
//	glm::vec3 normal_color = ray_pool[index].direction;
//	normal_color.x = ( normal_color.x < 0.0f ) ? ( normal_color.x * -1.0f ) : normal_color.x;
//	normal_color.y = ( normal_color.y < 0.0f ) ? ( normal_color.y * -1.0f ) : normal_color.y;
//	normal_color.z = ( normal_color.z < 0.0f ) ? ( normal_color.z * -1.0f ) : normal_color.z;
//
//	image[index] = normal_color;
//}


//__global__
//void uselessKernel()
//{
//	int ray_pool_index = ( blockIdx.x * blockDim.x ) + threadIdx.x;
//	return;
//}


// Core raytracer kernel.
__global__
void raytraceRay( ray *ray_pool,
				  int ray_pool_size,
				  glm::vec2 resolution,
				  float current_iteration, // Used solely for random number generation (I think).
				  cameraData cam,
				  int raytrace_depth,
				  glm::vec3 *image,
				  staticGeom *geoms,
				  int num_geoms,
				  material *materials )
{
	int ray_pool_index = ( blockIdx.x * blockDim.x ) + threadIdx.x;

	if ( ray_pool_index > ray_pool_size ) {
		return;
	}

	ray r = ray_pool[ray_pool_index];
	int image_pixel_index = ( r.image_coords.y * ( int )resolution.x ) + r.image_coords.x;

	// Nudge ray along it's direction to avoid intersecting with the surface it originates from.
	r.origin += ( r.direction * 0.001f );

	// Intersection testing.
	float dist_to_intersection;
	int material_index;
	glm::vec3 intersection_point;
	glm::vec3 intersection_normal;
	bool ray_did_intersect_something = sceneIntersection( r,						// Current ray.
														  geoms,					// List of scene geometry.
														  num_geoms,				// Number of pieces of geometry.
														  dist_to_intersection,		// Reference to be filled.
														  material_index,			// Reference to be filled.
														  intersection_point,		// Reference to be filled.
														  intersection_normal );	// Reference to be filled.


	// Ray misses. Return background color. Kill ray.
	if ( !ray_did_intersect_something ) {
		image[image_pixel_index] += glm::vec3( 0.0f, 0.0f, 0.0f );
		r.is_active = false;
		ray_pool[ray_pool_index] = r;
		return;
	}

	// Get material of intersected object.
	material mat = materials[material_index];

	// Use Roussian Roulette to randomly kill the current ray.
	glm::vec3 f = mat.color;
	float p = ( f.x > f.y && f.x > f.z ) ? f.x : ( ( f.y > f.z ) ? f.y : f.z );
    if ( raytrace_depth > 5 ) {
		glm::vec3 rand = generateRandomNumberFromThread( resolution, ( current_iteration * raytrace_depth ), r.image_coords.x, r.image_coords.y );
        if ( rand.x < p ) {
            f = f * ( 1.0f / p );
        }
        else {
			image[image_pixel_index] += ( r.color * f * mat.emittance );
			r.is_active = false;
			ray_pool[ray_pool_index] = r;
			return;
        }
    }

	// Ray hits light source. Add acculumated color contribution of ray. Kill ray.
	if ( mat.emittance > 0.0f ) {
		image[image_pixel_index] += ( r.color * f * mat.emittance );
		r.is_active = false;
		ray_pool[ray_pool_index] = r;
		return;
	}

	MaterialType mat_type = determineMaterialType( mat );
	if ( mat_type == IDEAL_DIFFUSE ) {
		glm::vec3 rand = generateRandomNumberFromThread( resolution,
														 current_iteration * raytrace_depth,
														 r.image_coords.x,
														 r.image_coords.y );
		r.direction = calculateRandomDirectionInHemisphere( intersection_normal,
															rand.x,
															rand.y );

		//// Compute direct illumination contribution.
		//glm::vec3 e( 0.0f, 0.0f, 0.0f );
		//for ( int i = 0; i < num_geoms; ++i ) {
		//	const staticGeom &s = geoms[i];
		//	material l_mat = materials[s.materialid];

		//	// Skip geometry that isn't a light source.
		//	if ( l_mat.emittance < 0.0f ) {
		//		continue;
		//	}

		//	glm::vec3 light_dir;
		//	if ( s.type == SPHERE ) {
		//		light_dir = glm::normalize( getRandomPointOnSphere( s, ( current_iteration * raytrace_depth ) ) - intersection_point );
		//	}
		//	else if ( s.type == CUBE ) {
		//		light_dir = glm::normalize( getRandomPointOnCube( s, ( current_iteration * raytrace_depth ) ) - intersection_point );
		//	}
		//	else if ( s.type == MESH ) {
		//		// TODO.
		//	}
		//	else {
		//		// ERROR: Unrecognized geometry type.
		//	}

		//	ray light_ray;
		//	light_ray.direction = light_dir;
		//	light_ray.origin = intersection_point;

		//	// Intersection testing.
		//	int light_id;
		//	bool did_hit_light = lightIntersection( light_ray, geoms, num_geoms, light_id );

		//	glm::vec3 intersection_normal_oriented = ( glm::dot( intersection_normal, r.direction ) < 0.0f ) ? intersection_normal : ( -1.0f * intersection_normal );

		//	if ( did_hit_light && light_id == i ) {
		//		e += f * ( l_mat.emittance * glm::dot( light_dir, intersection_normal_oriented ) );
		//	}
		//}

		r.color = r.color * f;			// Only add color contributions of this ray if it makes contact with a light source.
		r.origin = intersection_point;	// Set origin point for next ray.
		ray_pool[ray_pool_index] = r;	// Update ray in ray pool.
		return;
	}
	else if ( mat_type == PERFECT_SPUCULAR ) {
		r.direction = calculateReflectionDirection( intersection_normal, r.direction );	// Mirror surface contributes no color.
		r.origin = intersection_point;													// Set origin point for next ray.
		ray_pool[ray_pool_index] = r;													// Update ray in ray pool.
		return;
	}
	else if ( mat_type == GLASS ) {
		glm::vec3 intersection_normal_oriented = ( glm::dot( intersection_normal, r.direction ) < 0.0f ) ? intersection_normal : ( -1.0f * intersection_normal );
		bool ray_is_entering = ( glm::dot( intersection_normal, intersection_normal_oriented ) > 0.0f );

		const float IOR_GLASS = 1.5f;
		float ior_incident = ray_is_entering ? 1.0f : IOR_GLASS;
		float ior_transmitted = ray_is_entering ? IOR_GLASS : 1.0f;

		glm::vec3 refl_dir = calculateReflectionDirection( intersection_normal, r.direction );
		glm::vec3 trans_dir = calculateTransmissionDirection( intersection_normal, r.direction, ior_incident, ior_transmitted );

		Fresnel f = calculateFresnel( intersection_normal, r.direction, ior_incident, ior_transmitted, refl_dir, trans_dir );

		glm::vec3 rand = generateRandomNumberFromThread( resolution, ( current_iteration * raytrace_depth ), r.image_coords.x, r.image_coords.y );

		if ( rand.x < f.reflectionCoefficient ) {
			r.direction = refl_dir;
			r.origin = intersection_point;
			ray_pool[ray_pool_index] = r;
			return;
		}
		else {
			r.direction = trans_dir;
			r.origin = intersection_point;
			ray_pool[ray_pool_index] = r;
			return;
		}
	}
}


__host__
__device__
MaterialType determineMaterialType( material mat )
{
	if ( mat.hasRefractive ) {
		return GLASS;
	}
	else if ( mat.hasReflective ) {
		return PERFECT_SPUCULAR;
	}
	else {
		return IDEAL_DIFFUSE;
	}
}



// thrust predicate to cull inactive rays from ray pool.
struct RayIsInactive
{
	__host__
	__device__
	bool operator()( const ray &r )
	{
		return !r.is_active;
	}
};


// Wrapper that sets up kernel calls and handles memory management.
// Handles one pathtrace iteration. Called many times to produce a rendered image.
void cudaRaytraceCore( uchar4 *pbo_pos,
					   camera *render_cam,
					   int frame,
					   int current_iteration,
					   material *materials,
					   int num_materials,
					   geom *geoms,
					   int num_geoms )
{
	// Tune these for performance.
	int depth = 10;
	int camera_raycast_tile_size = 8;
	int raytrace_tile_size = 128;

	// Setup crucial magic.
	dim3 threads_per_block( camera_raycast_tile_size,
							camera_raycast_tile_size );
	dim3 full_blocks_per_grid( ( int )ceil( ( float )render_cam->resolution.x / ( float )camera_raycast_tile_size ),
							   ( int )ceil( ( float )render_cam->resolution.y / ( float )camera_raycast_tile_size ) );
  
	// Send image to GPU.
	glm::vec3 *cuda_image = NULL;
	float size_image = ( int )render_cam->resolution.x * ( int )render_cam->resolution.y * sizeof( glm::vec3 );
	hipMalloc( ( void** )&cuda_image,
				size_image );
	hipMemcpy( cuda_image,
				render_cam->image,
				size_image,
				hipMemcpyHostToDevice );
  
	// Package up geometry.
	staticGeom *geom_list = new staticGeom[num_geoms];
	for ( int i = 0; i < num_geoms; ++i ) {
		staticGeom newStaticGeom;
		newStaticGeom.type = geoms[i].type;
		newStaticGeom.materialid = geoms[i].materialid;
		newStaticGeom.translation = geoms[i].translations[frame];
		newStaticGeom.rotation = geoms[i].rotations[frame];
		newStaticGeom.scale = geoms[i].scales[frame];
		newStaticGeom.transform = geoms[i].transforms[frame];
		newStaticGeom.inverseTransform = geoms[i].inverseTransforms[frame];
		geom_list[i] = newStaticGeom;
	}
  
	// Send geometry to GPU.
	staticGeom *cuda_geoms = NULL;
	float size_geom_list = num_geoms * sizeof( staticGeom );
	hipMalloc( ( void** )&cuda_geoms,
				size_geom_list );
	hipMemcpy( cuda_geoms,
				geom_list,
				size_geom_list,
				hipMemcpyHostToDevice );
  
	// Send materials to GPU.
	material *cuda_materials = NULL;
	float size_material_list = num_materials * sizeof( material );
	hipMalloc( ( void** )&cuda_materials,
				size_material_list );
	hipMemcpy( cuda_materials,
				materials,
				size_material_list,
				hipMemcpyHostToDevice );
  
	// Package up camera.
	cameraData cam;
	cam.resolution = render_cam->resolution;
	cam.position = render_cam->positions[frame];
	cam.view = render_cam->views[frame];
	cam.up = render_cam->ups[frame];
	cam.fov = render_cam->fov;
	
	// Variables to compute rays originating from render camera.
	glm::vec3 a = glm::cross( cam.view, cam.up );
	glm::vec3 m = cam.position + cam.view; // Midpoint of frame buffer.
	glm::vec3 h = ( a * glm::length( cam.view ) * ( float )tan( cam.fov.x * ( PI / 180.0f ) ) ) / glm::length( a ); // Horizontal NDC value.
	glm::vec3 v = glm::vec3( 0.0f, cam.resolution.y * glm::length( h ) / cam.resolution.x, 0.0f ); // Vertical NDC value.

	// Allocate device memory for ray pool.
	ray *cuda_ray_pool = NULL;
	int num_rays = ( int )( render_cam->resolution.x * render_cam->resolution.y );
	hipMalloc( ( void** )&cuda_ray_pool,
				num_rays * sizeof( ray ) );
	
	// TODO: Mod current iteration # with pixel subsample # for super-sampled anti-aliasing.

	// Initialize ray pool with rays originating at the render camera directed through each pixel in the image buffer.
	raycastFromCameraKernel<<< full_blocks_per_grid, threads_per_block >>>( cuda_ray_pool,
																			render_cam->resolution,
																			cam.position,
																			m,
																			h,
																			v,
																			( float )current_iteration );

	//testOutputKernel<<< full_blocks_per_grid, threads_per_block >>>( cuda_image,
	//																 cuda_ray_pool,
	//																 render_cam->resolution );

	// Launch raytraceRay kernel once per raytrace depth.
	for ( int i = 0; i < depth; ++i ) {
		dim3 threads_per_raytrace_block( raytrace_tile_size );
		dim3 blocks_per_raytrace_grid( ( int )ceil( ( float )num_rays / ( float )raytrace_tile_size ) );

		// Test.
		//uselessKernel<<< blocks_per_raytrace_grid, threads_per_raytrace_block >>>();

		// Launch raytraceRay kernel.
		raytraceRay<<< blocks_per_raytrace_grid, threads_per_raytrace_block >>>( cuda_ray_pool,
																				 num_rays,
																				 render_cam->resolution,
																				 ( float )current_iteration,
																				 cam,
																				 ( i + 1 ),
																				 cuda_image,
																				 cuda_geoms,
																				 num_geoms,
																				 cuda_materials );

		// Note: Stream compaction is slow.
		thrust::device_ptr<ray> ray_pool_device_ptr( cuda_ray_pool );
		thrust::device_ptr<ray> culled_ray_pool_device_ptr = thrust::remove_if( ray_pool_device_ptr,
																				ray_pool_device_ptr + num_rays,
																				RayIsInactive() );

		// Compute number of active rays in ray pool.
		num_rays = culled_ray_pool_device_ptr.get() - ray_pool_device_ptr.get();
	}

	// Launch sendImageToPBO kernel.
	sendImageToPBO<<< full_blocks_per_grid, threads_per_block >>>( pbo_pos,
																   render_cam->resolution,
																   cuda_image,
																   current_iteration );

	// Retrieve image from GPU.
	hipMemcpy( render_cam->image,
				cuda_image,
				size_image,
				hipMemcpyDeviceToHost );

	// Cleanup.
	hipFree( cuda_image );
	hipFree( cuda_geoms );
	hipFree( cuda_materials );
	hipFree( cuda_ray_pool );
	delete geom_list;

	// Make certain the kernel has completed.
	//hipDeviceSynchronize(); // Deprecated.
	hipDeviceSynchronize();

	checkCUDAError( "Kernel failed!" );
}