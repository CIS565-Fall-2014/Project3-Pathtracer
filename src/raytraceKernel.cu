#include "hip/hip_runtime.h"
// CIS565 CUDA Raytracer: A parallel raytracer for Patrick Cozzi's CIS565: GPU Computing at the University of Pennsylvania
// Written by Yining Karl Li, Copyright (c) 2012 University of Pennsylvania
// This file includes code from:
//       Rob Farber for CUDA-GL interop, from CUDA Supercomputing For The Masses: http://www.drdobbs.com/architecture-and-design/cuda-supercomputing-for-the-masses-part/222600097
//       Peter Kutz and Yining Karl Li's GPU Pathtracer: http://gpupathtracer.blogspot.com/
//       Yining Karl Li's TAKUA Render, a massively parallel pathtracing renderer: http://www.yiningkarlli.com

#include <stdio.h>
#include <hip/hip_runtime.h>
#include <cmath>

#include "sceneStructs.h"
#include "glm/glm.hpp"
#include "utilities.h"
#include "raytraceKernel.h"
#include "intersections.h"
#include "interactions.h"

void checkCUDAError(const char *msg) {
  hipError_t err = hipGetLastError();
  if( hipSuccess != err) {
    fprintf(stderr, "Cuda error: %s: %s.\n", msg, hipGetErrorString( err) ); 
    exit(EXIT_FAILURE); 
  }
} 

// LOOK: This function demonstrates how to use thrust for random number generation on the GPU!
// Function that generates static.
__host__ __device__ glm::vec3 generateRandomNumberFromThread(glm::vec2 resolution, float time, int x, int y){
  int index = x + (y * resolution.x);
   
  thrust::default_random_engine rng(hash(index*time));
  thrust::uniform_real_distribution<float> u01(0,1);

  return glm::vec3((float) u01(rng), (float) u01(rng), (float) u01(rng));
}

// TODO: IMPLEMENT THIS FUNCTION
// Function that does the initial raycast from the camera
__host__ __device__ ray raycastFromCameraKernel(glm::vec2 resolution, float time, int x, int y, glm::vec3 eye, glm::vec3 view, glm::vec3 up, glm::vec2 fov){
	ray r;
	//r.origin = glm::vec3(0,0,0);
	//r.direction = glm::vec3(0,0,-1);

	view = glm::normalize(view);
	glm::vec3 vecA = glm::normalize(glm::cross(view, up));// center to right
	glm::vec3 vecB = glm::normalize(glm::cross(vecA, view));// center to up

	glm::vec3 vecV = vecB * glm::length(view) * (float)tan(fov.y/ 180.0f * PI) * (1.0f - (1.0f + y * 2.0f) /resolution.y);
	glm::vec3 vecH = vecA * glm::length(view) * (float)tan(fov.x/ 180.0f * PI) * (1.0f - (1.0f + x * 2.0f) /resolution.x);

	glm::vec3 rayDir = glm::normalize(view + vecV + vecH);

	r.origin = eye;
	r.direction = rayDir;


	return r;
}

__host__ __device__ glm::vec3 getSpecularColor(ray* light, int lightCount, glm::vec3* lightColor, ray r, glm::vec3 faceNormal, float specularExp){
	//float specularColorR = 0;
	//float specularColorG = 0;
	//float specularColorB = 0;
	glm::vec3 specularColor(0,0,0);

	for(int i = 0; i < lightCount ; i++)
	{
		glm::vec3 reflectLight = -1.0f * glm::normalize( light[i].direction - faceNormal * 2.0f * glm::dot(light[i].direction, faceNormal));
		//reflectLight = -1.0f * reflectLight / sqrt(dot(reflectLight, reflectLight));

		//float alpha = acos(glm::dot(r.direction, reflectLight));
		//float specularTerm = pow(cos(alpha), specularExp);
		float specularTerm = 0.0f;
		float dotProduct = glm::dot(r.direction, reflectLight);
		if(dotProduct < 0)
			specularTerm = 0;
		else{
			specularTerm = pow(glm::dot(r.direction, reflectLight), specularExp);
		}
		//if(specularTerm < 0.0)
		//	specularTerm = 0.0f;
		//else if(specularTerm > 1.0)
		//	specularTerm = 1.0f;

		//specularColorR += specularTerm * lightColor[i].x;
		//specularColorG += specularTerm * lightColor[i].y;
		//specularColorB += specularTerm * lightColor[i].z;

		specularColor += specularTerm * lightColor[i];
	}
	
	//return glm::vec3(specularColorR, specularColorG, specularColorB);
	return specularColor;
}

//__host__ __device__ glm::vec3 getDiffuseColor(ray* light, int lightCount, glm::vec3* lightColor, ray r, glm::vec3 faceNormal){
//	return glm::vec3(0,0,0);
//}

__host__ __device__ glm::vec3 raytraceRecursive(ray r, int depth, /*glm::vec3* lightPos, int lightCount,*/ material* materials, int numberOfMaterials, staticGeom* geoms, int numberOfGeoms){

	if(depth <= 0)
		return glm::vec3(0,0,0);

	bool hitCheck = false;
	float shortestDis = -1;
	int hitObjectIndex = -1;
	glm::vec3 intersectionPoint(0,0,0);
	glm::vec3 intersectionNormal(0,0,0);

	for(int i = 0; i < numberOfGeoms; ++i){
		float dis = -1;
		glm::vec3 objIntersectPt(0, 0, 0);
		glm::vec3 objIntersectN(0, 0, 0);
		switch(geoms[i].type){
			case SPHERE:
				dis = sphereIntersectionTest(geoms[i], r, objIntersectPt, objIntersectN);
				break;
			case CUBE:
				dis = boxIntersectionTest(geoms[i], r, objIntersectPt, objIntersectN);
				break;
			case MESH:
				break;
		}

		if((dis != -1 && shortestDis == -1) || (dis != -1 && shortestDis != -1 && dis < shortestDis && dis > 0)){
			hitCheck = true;
			shortestDis = dis;
			intersectionPoint = objIntersectPt;
			intersectionNormal = objIntersectN;
			hitObjectIndex = i;


		}
	}
	if(hitCheck == false){


		return glm::vec3(0,0,0);
	}
	else{
		material mate = materials[hitObjectIndex];

		
		if(mate.emittance != 0){ //hit light, so terminate the ray
			return mate.color * mate.emittance / 5.0f;
		}

		ray newRay;
		newRay.origin = glm::vec3(0,0,0);
		newRay.direction = glm::vec3(0,0,0);



		//glm::vec3 newEyePositionOut = intersectionPoint - r.direction * (float)EPSILON;//���@��epsloon�קKray���i�hface�̭�
		//glm::vec3 newEyePositionIn = intersectionPoint + r.direction * (float)EPSILON;//���@��epsloon��ray���i�hface�̭�

		//glm::vec3* light2HitPtArray = new glm::vec3[lightCount];
		//float* disLight2HitPtArray = new float[lightCount];

		//for(int i = 0 ; i < lightCount ; i++){
		//	light2HitPtArray[i] = glm::normalize(newEyePositionOut - lightPos[i]);
		//	disLight2HitPtArray[i] = glm::length(light2HitPtArray[i]);
		//}


		//Reflect Color
		glm::vec3 reflectColor;
		if(mate.hasReflective != 0)//TODO  Check if really use this attribute
			reflectColor = raytraceRecursive(newRay, --depth, /*lightPos, lightCount,*/ materials, numberOfMaterials, geoms, numberOfGeoms);
		else
			reflectColor = glm::vec3(0, 0, 0);

		//Refract Color


		//Diffuse Color


		//Specular Color

		//getSpecularColor(ray* light, int lightCount, glm::vec3* lightColor, ray r, glm::vec3 faceNormal, float specularExp);

		glm::vec3 currentPtColor = reflectColor + mate.color;
		return currentPtColor;
	}


	return glm::vec3(0,0,0);
}

//Kernel that blacks out a given image buffer
__global__ void clearImage(glm::vec2 resolution, glm::vec3* image){
    int x = (blockIdx.x * blockDim.x) + threadIdx.x;
    int y = (blockIdx.y * blockDim.y) + threadIdx.y;
    int index = x + (y * resolution.x);
    if(x<=resolution.x && y<=resolution.y){
      image[index] = glm::vec3(0,0,0);
    }
}

//Kernel that writes the image to the OpenGL PBO directly.
__global__ void sendImageToPBO(uchar4* PBOpos, glm::vec2 resolution, glm::vec3* image){
  
  int x = (blockIdx.x * blockDim.x) + threadIdx.x;
  int y = (blockIdx.y * blockDim.y) + threadIdx.y;
  int index = x + (y * resolution.x);
  
  if(x<=resolution.x && y<=resolution.y){

      glm::vec3 color;
      color.x = image[index].x*255.0;
      color.y = image[index].y*255.0;
      color.z = image[index].z*255.0;

      if(color.x>255){
        color.x = 255;
      }

      if(color.y>255){
        color.y = 255;
      }

      if(color.z>255){
        color.z = 255;
      }
      
      // Each thread writes one pixel location in the texture (textel)
      PBOpos[index].w = 0;
      PBOpos[index].x = color.x;
      PBOpos[index].y = color.y;
      PBOpos[index].z = color.z;
  }
}

// TODO: IMPLEMENT THIS FUNCTION
// Core raytracer kernel
__global__ void raytraceRay(glm::vec2 resolution, float time, cameraData cam, int rayDepth, glm::vec3* colors,
                            material* materials, int numberOfMaterials, staticGeom* geoms, int numberOfGeoms){
	
	//int lightCount = 1;
	//glm::vec3* lightPosArray = new glm::vec3[lightCount];
	//lightPosArray[0] = glm::vec3(0,0,0);

	int x = (blockIdx.x * blockDim.x) + threadIdx.x;
	int y = (blockIdx.y * blockDim.y) + threadIdx.y;
	int index = x + (y * resolution.x);
	glm::vec3 pixelColor(0, 0, 0);
	if((x < resolution.x && y < resolution.y )){
		ray r = raycastFromCameraKernel(resolution, time, x, y, cam.position, cam.view, cam.up, cam.fov);
		
			
		glm::vec3 newColorEnergy =raytraceRecursive(r, rayDepth, /*lightPosArray, lightCount,*/ materials, numberOfMaterials, geoms, numberOfGeoms);
		glm::vec3 oldColorEnergy = colors[index] * (time - 1);
		glm::vec3 newColor = (newColorEnergy + oldColorEnergy) / time;
		colors[index] = newColor;

		 //glm::vec3 colorReflect = glm::vec3(0,0,0);;// = raytraceRay(resolution, time, ;

		//colors[index] = colorBRDF + colorReflect;

	}


}

// TODO: FINISH THIS FUNCTION
// Wrapper for the __global__ call that sets up the kernel calls and does a ton of memory management
void cudaRaytraceCore(uchar4* PBOpos, camera* renderCam, int frame, int iterations, material* materials, int numberOfMaterials, geom* geoms, int numberOfGeoms){
  
  int traceDepth = 1; //determines how many bounces the raytracer traces

  // set up crucial magic
  int tileSize = 8;
  dim3 threadsPerBlock(tileSize, tileSize);
  dim3 fullBlocksPerGrid((int)ceil(float(renderCam->resolution.x)/float(tileSize)), (int)ceil(float(renderCam->resolution.y)/float(tileSize)));
  
  // send image to GPU
  glm::vec3* cudaimage = NULL;
  hipMalloc((void**)&cudaimage, (int)renderCam->resolution.x*(int)renderCam->resolution.y*sizeof(glm::vec3));
  hipMemcpy( cudaimage, renderCam->image, (int)renderCam->resolution.x*(int)renderCam->resolution.y*sizeof(glm::vec3), hipMemcpyHostToDevice);
  
  // package geometry and materials and sent to GPU
  staticGeom* geomList = new staticGeom[numberOfGeoms];
  for(int i=0; i<numberOfGeoms; i++){
    staticGeom newStaticGeom;
    newStaticGeom.type = geoms[i].type;
    newStaticGeom.materialid = geoms[i].materialid;
    newStaticGeom.translation = geoms[i].translations[frame];
    newStaticGeom.rotation = geoms[i].rotations[frame];
    newStaticGeom.scale = geoms[i].scales[frame];
    newStaticGeom.transform = geoms[i].transforms[frame];
    newStaticGeom.inverseTransform = geoms[i].inverseTransforms[frame];
    geomList[i] = newStaticGeom;
  }
  
  staticGeom* cudageoms = NULL;
  hipMalloc((void**)&cudageoms, numberOfGeoms*sizeof(staticGeom));
  hipMemcpy( cudageoms, geomList, numberOfGeoms*sizeof(staticGeom), hipMemcpyHostToDevice);
  
  material* cudaMaterials = NULL;
  hipMalloc((void**)&cudaMaterials, numberOfMaterials*sizeof(material));
  hipMemcpy( cudaMaterials, materials, numberOfMaterials*sizeof(material), hipMemcpyHostToDevice);




  // package camera
  cameraData cam;
  cam.resolution = renderCam->resolution;
  cam.position = renderCam->positions[frame];
  cam.view = renderCam->views[frame];
  cam.up = renderCam->ups[frame];
  cam.fov = renderCam->fov;

  // kernel launches
  raytraceRay<<<fullBlocksPerGrid, threadsPerBlock>>>(renderCam->resolution, (float)iterations, cam, traceDepth, cudaimage, cudaMaterials, numberOfMaterials, cudageoms, numberOfGeoms);

  sendImageToPBO<<<fullBlocksPerGrid, threadsPerBlock>>>(PBOpos, renderCam->resolution, cudaimage);

  // retrieve image from GPU
  hipMemcpy( renderCam->image, cudaimage, (int)renderCam->resolution.x*(int)renderCam->resolution.y*sizeof(glm::vec3), hipMemcpyDeviceToHost);

  // free up stuff, or else we'll leak memory like a madman
  hipFree( cudaimage );
  hipFree( cudageoms );
  delete geomList;

  // make certain the kernel has completed
  hipDeviceSynchronize();

  checkCUDAError("Kernel failed!");
}
