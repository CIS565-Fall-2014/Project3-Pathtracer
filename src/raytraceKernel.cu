#include "hip/hip_runtime.h"
// CIS565 CUDA Raytracer: A parallel raytracer for Patrick Cozzi's CIS565: GPU Computing at the University of Pennsylvania
// Written by Yining Karl Li, Copyright (c) 2012 University of Pennsylvania
// This file includes code from:
//       Rob Farber for CUDA-GL interop, from CUDA Supercomputing For The Masses: http://www.drdobbs.com/architecture-and-design/cuda-supercomputing-for-the-masses-part/222600097
//       Peter Kutz and Yining Karl Li's GPU Pathtracer: http://gpupathtracer.blogspot.com/
//       Yining Karl Li's TAKUA Render, a massively parallel pathtracing renderer: http://www.yiningkarlli.com

// SEE HANDWRITTEN NOTES TO UNDERSTAND WHAT TO DO!

#include <stdio.h>
#include <hip/hip_runtime.h>
#include <cmath>

#include "sceneStructs.h"
#include "glm/glm.hpp"
#include "utilities.h"
#include "raytraceKernel.h"
#include "intersections.h"
#include "interactions.h"




#define TRACE_DEPTH_LIMIT 5




void checkCUDAError(const char *msg) {
  hipError_t err = hipGetLastError();
  if( hipSuccess != err) {
    fprintf(stderr, "Cuda error: %s: %s.\n", msg, hipGetErrorString( err) ); 
    exit(EXIT_FAILURE); 
  }
} 

// LOOK: This function demonstrates how to use thrust for random number generation on the GPU!
// Function that generates static.
__host__ __device__ glm::vec3 generateRandomNumberFromThread(glm::vec2 resolution, float time, int x, int y){
  int index = x + (y * resolution.x);
   
  thrust::default_random_engine rng(hash(index*time));
  thrust::uniform_real_distribution<float> u01(0,1);

  return glm::vec3((float) u01(rng), (float) u01(rng), (float) u01(rng));
}

// TODO: IMPLEMENT THIS FUNCTION
// Function that does the initial raycast from the camera

// HINT: look at the first homework from CIS560! Start at line 250 and go down through the for loop at 328: all
//       the mathematics I need is already there!
//       "time" seems to refer to iteration number. Probably useful for a depth-of-field effect or something, but I will ignore it for now.
//       These rays could easily be "saved" to provide further optimization (so they aren't recalculated with each iteration), but in case
//       a DOF effect is implemented later on and "jittering" is required, I'll overlook this potential optimization.
__host__ __device__ ray raycastFromCameraKernel(glm::vec2 resolution, float time, int x, int y, glm::vec3 eye, glm::vec3 view, glm::vec3 up, glm::vec2 fov){

	// the HORIZONAL direction of the viewing plane, calculated with the "up" vector
	glm::vec3 A = glm::cross(view, up);
	// the VERTICAL direction of the viewing plane
	glm::vec3 B = glm::cross(A, view);

	// central point on the image plane that vectors from the eye are being drawn towards
	glm::vec3 M = eye + view;

	float phi = fov.y;
	float theta = fov.x;

	//rescaled HORIZONTAL
	glm::vec3 H = A * glm::length(view) * tan(theta) / glm::length(A);
	//rescaled VERTICAL
	glm::vec3 V = B * glm::length(view) * tan(phi) / glm::length(B);

	float sx = (float) x / (float) (resolution.x - 1);
	float sy = (float) y / (float) (resolution.y - 1);

	glm::vec3 screenPoint = M + (2*sx-1) * H + (2*sy-1) * V;

	ray r;
	r.direction = glm::normalize(screenPoint - eye);
	r.origin = eye;
	r.active = true;
	r.sourceindex = x + (y * resolution.x);
	r.color = glm::vec3(0,0,0);
	r.intensityMultiplier = 1.f;

	return r;
}

//Kernel that blacks out a given image buffer
__global__ void clearImage(glm::vec2 resolution, glm::vec3* image){
    int x = (blockIdx.x * blockDim.x) + threadIdx.x;
    int y = (blockIdx.y * blockDim.y) + threadIdx.y;
    int index = x + (y * resolution.x);
    if(x<=resolution.x && y<=resolution.y){
      image[index] = glm::vec3(0,0,0);
    }
}

//Kernel that writes the image to the OpenGL PBO directly.
__global__ void sendImageToPBO(uchar4* PBOpos, glm::vec2 resolution, glm::vec3* image, float time){
  
  int x = (blockIdx.x * blockDim.x) + threadIdx.x;
  int y = (blockIdx.y * blockDim.y) + threadIdx.y;
  int index = x + (y * resolution.x);
  
  if(x<=resolution.x && y<=resolution.y){

      glm::vec3 color;
      color.x = image[index].x*255.0;
      color.y = image[index].y*255.0;
      color.z = image[index].z*255.0;

	  // output needs to be normalized against number of iterations for EACH frame drawn to the screen
	  color.r /= time;
	  color.g /= time;
	  color.b /= time;

      if(color.x>255){
        color.x = 255;
      }

      if(color.y>255){
        color.y = 255;
      }

      if(color.z>255){
        color.z = 255;
      }
      
      // Each thread writes one pixel location in the texture (textel)
      PBOpos[index].w = 0;
      PBOpos[index].x = color.x;
      PBOpos[index].y = color.y;
      PBOpos[index].z = color.z;
  }
}

// TODO: IMPLEMENT THIS FUNCTION
// Core raytracer kernel

// NOTE: I believe I just need to ADD an argument or so for materials/lights (lights are just materials with emittance)

// NOTE: this kernel represents "tracing ONE bounce" 
__global__ void raytraceRay(glm::vec2 resolution, float time, cameraData cam, int rayDepth, glm::vec3* colors,
                            staticGeom* geoms, int numberOfGeoms){

  int x = (blockIdx.x * blockDim.x) + threadIdx.x;
  int y = (blockIdx.y * blockDim.y) + threadIdx.y;
  int index = x + (y * resolution.x);

  // REMEMBER:
  /*
       - do a colors[] +=, not a colors[] = ... I need to ACCUMULATE colors and then divide (see PBO function above)
	   - use the pooled array map to ensure you're +='ing to the proper colors[] entry!
  */
  if((x<=resolution.x && y<=resolution.y)){
	  colors[index] += generateRandomNumberFromThread(resolution, time, x, y);
  }
}

// TODO: FINISH THIS FUNCTION ("Support passing materials and lights to CUDA")
// Wrapper for the __global__ call that sets up the kernel calls and does a ton of memory management

/*
	 - This function manages an array of "pooled" rays which provides stream compaction optimization.
*/
void cudaRaytraceCore(uchar4* PBOpos, camera* renderCam, int frame, int iterations, material* materials, int numberOfMaterials, geom* geoms, int numberOfGeoms){
  
  int traceDepth = 0; //determines how many bounces the raytracer traces

  // set up crucial magic
  int tileSize = 8;
  dim3 threadsPerBlock(tileSize, tileSize);
  dim3 fullBlocksPerGrid((int)ceil(float(renderCam->resolution.x)/float(tileSize)), (int)ceil(float(renderCam->resolution.y)/float(tileSize)));
  
  // send image to GPU
  glm::vec3* cudaimage = NULL;
  hipMalloc((void**)&cudaimage, (int)renderCam->resolution.x*(int)renderCam->resolution.y*sizeof(glm::vec3));
  hipMemcpy( cudaimage, renderCam->image, (int)renderCam->resolution.x*(int)renderCam->resolution.y*sizeof(glm::vec3), hipMemcpyHostToDevice);
  
  // package geometry and materials and sent to GPU
  staticGeom* geomList = new staticGeom[numberOfGeoms];
  for(int i=0; i<numberOfGeoms; i++){
    staticGeom newStaticGeom;
    newStaticGeom.type = geoms[i].type;
    newStaticGeom.materialid = geoms[i].materialid;
    newStaticGeom.translation = geoms[i].translations[frame];
    newStaticGeom.rotation = geoms[i].rotations[frame];
    newStaticGeom.scale = geoms[i].scales[frame];
    newStaticGeom.transform = geoms[i].transforms[frame];
    newStaticGeom.inverseTransform = geoms[i].inverseTransforms[frame];
    geomList[i] = newStaticGeom;
  }
  
  staticGeom* cudageoms = NULL;
  hipMalloc((void**)&cudageoms, numberOfGeoms*sizeof(staticGeom));
  hipMemcpy( cudageoms, geomList, numberOfGeoms*sizeof(staticGeom), hipMemcpyHostToDevice);
  
  // TODO: package materials/lights and send to CUDA

  // package camera
  cameraData cam;
  cam.resolution = renderCam->resolution;
  cam.position = renderCam->positions[frame];
  cam.view = renderCam->views[frame];
  cam.up = renderCam->ups[frame];
  cam.fov = renderCam->fov;

  //std::cout << "\nKernel launches about to start\n" << std::endl;
  //std::cout << "iterations is: " << iterations << std::endl;
  // kernel launches
  raytraceRay<<<fullBlocksPerGrid, threadsPerBlock>>>(renderCam->resolution, (float)iterations, cam, traceDepth, cudaimage, cudageoms, numberOfGeoms);

  //std::cout << "\nraytraceRay call is done\n" << std::endl;

  sendImageToPBO<<<fullBlocksPerGrid, threadsPerBlock>>>(PBOpos, renderCam->resolution, cudaimage, (float)iterations);

  //std::cout << "\nKernel calls are done\n" << std::endl;

  // retrieve image from GPU
  hipMemcpy( renderCam->image, cudaimage, (int)renderCam->resolution.x*(int)renderCam->resolution.y*sizeof(glm::vec3), hipMemcpyDeviceToHost);

  // free up stuff, or else we'll leak memory like a madman
  hipFree( cudaimage );
  hipFree( cudageoms );
  delete geomList;

  // make certain the kernel has completed
  hipDeviceSynchronize();

  checkCUDAError("Kernel failed!");
}
