#include "hip/hip_runtime.h"
// CIS565 CUDA Raytracer: A parallel raytracer for Patrick Cozzi's CIS565: GPU Computing at the University of Pennsylvania
// Written by Yining Karl Li, Copyright (c) 2012 University of Pennsylvania
// This file includes code from:
//       Rob Farber for CUDA-GL interop, from CUDA Supercomputing For The Masses: http://www.drdobbs.com/architecture-and-design/cuda-supercomputing-for-the-masses-part/222600097
//       Peter Kutz and Yining Karl Li's GPU Pathtracer: http://gpupathtracer.blogspot.com/
//       Yining Karl Li's TAKUA Render, a massively parallel pathtracing renderer: http://www.yiningkarlli.com

#include <stdio.h>
#include <hip/hip_runtime.h>
#include <cmath>

#include <thrust/host_vector.h>
#include <thrust/device_vector.h>
#include <thrust/count.h>
#include <thrust/scan.h>


#include "sceneStructs.h"
#include "glm/glm.hpp"
#include "utilities.h"
#include "raytraceKernel.h"
#include "intersections.h"
#include "interactions.h"




void checkCUDAError(const char *msg) {
  hipError_t err = hipGetLastError();
  if( hipSuccess != err) {
    fprintf(stderr, "Cuda error: %s: %s.\n", msg, hipGetErrorString( err) ); 
    exit(EXIT_FAILURE); 
  }
} 

// LOOK: This function demonstrates how to use thrust for random number generation on the GPU!
// Function that generates static.
__host__ __device__ glm::vec3 generateRandomNumberFromThread(glm::vec2 resolution, float time, int x, int y){
  int index = x + (y * resolution.x);
   
  thrust::default_random_engine rng(hash(index*time));
  thrust::uniform_real_distribution<float> u01(0,1);

  return glm::vec3((float) u01(rng), (float) u01(rng), (float) u01(rng));
}



///////////////////////////////////
//////////////////////////////////
// TODO: IMPLEMENT THIS FUNCTION/
////////////////////////////////
///////////////////////////////
// Function that does the initial raycast from the camera
__host__ __device__ ray raycastFromCameraKernel(glm::vec2 resolution, float time, int x, int y, glm::vec3 eye, glm::vec3 view, glm::vec3 up, glm::vec2 fov){
  int index = x + (y * resolution.x);
  glm::vec3 alpha, beta, midPix, horizScale, vertScale, pixel;
  alpha  = glm::cross(view, up);
  beta   = glm::cross(alpha, view);
  midPix = eye + view;

  vertScale  = glm::normalize(beta)  * glm::length(view) * tan(glm::radians( - fov.y)); //had to flip this (it was upside down)
  horizScale = glm::normalize(alpha) * glm::length(view) * tan(glm::radians(fov.x));
  
  //jitter the pixel
  thrust::default_random_engine rng(hash(index*time));
  thrust::uniform_real_distribution<float> u01(-0.5,0.5);
  thrust::uniform_real_distribution<float> u02(-0.01,0.01);
  
  pixel = midPix + horizScale * (float)((2.0 * (x + (float)u01(rng))/resolution.x) - 1.0) + vertScale * (float)((2.0 * (y + (float)u01(rng))/resolution.y) - 1.0);
  ray r;
  r.origin = eye;
  r.origin.x += (float)u02(rng);
  r.origin.y += (float)u02(rng);
  r.origin.z += (float)u02(rng);
  r.direction = glm::normalize(pixel - eye);
  return r;
}

//Kernel that blacks out a given image buffer
__global__ void clearImage(glm::vec2 resolution, glm::vec3* image){
    int x = (blockIdx.x * blockDim.x) + threadIdx.x;
    int y = (blockIdx.y * blockDim.y) + threadIdx.y;
    int index = x + (y * resolution.x);
    if(x<=resolution.x && y<=resolution.y){
      image[index] = glm::vec3(0,0,0);
    }
}

//Kernel that writes the image to the OpenGL PBO directly.
__global__ void sendImageToPBO(uchar4* PBOpos, glm::vec2 resolution, glm::vec3* image){
  
  int x = (blockIdx.x * blockDim.x) + threadIdx.x;
  int y = (blockIdx.y * blockDim.y) + threadIdx.y;
  int index = x + (y * resolution.x);
  
  if(x<=resolution.x && y<=resolution.y){

      glm::vec3 color;
      color.x = image[index].x*255.0;
      color.y = image[index].y*255.0;
      color.z = image[index].z*255.0;

      if(color.x>255){
        color.x = 255;
      }

      if(color.y>255){
        color.y = 255;
      }

      if(color.z>255){
        color.z = 255;
      }
      
      // Each thread writes one pixel location in the texture (textel)
      PBOpos[index].w = 0;
      PBOpos[index].x = color.x;
      PBOpos[index].y = color.y;
      PBOpos[index].z = color.z;
  }
}

//Initialize rays
__global__ void initializeRay(glm::vec2 resolution, float time, cameraData cam, rayState* rayList){
  int x = (blockIdx.x * blockDim.x) + threadIdx.x;
  int y = (blockIdx.y * blockDim.y) + threadIdx.y;
  int index = x + (y * resolution.x);

  if((x<=resolution.x && y<=resolution.y)){
    ray thisRay = raycastFromCameraKernel(resolution, time, x, y, cam.position, cam.view, cam.up, cam.fov);
    rayList[index].RAY      = thisRay;
    rayList[index].isValid  = 1;
    rayList[index].color    = glm::vec3(1,1,1);
    rayList[index].photoIDX = index;
  }
}



///////////////////////////////////
//////////////////////////////////
// TODO: IMPLEMENT THIS FUNCTION/ 
//   raytraceRay() should take in a camera, image buffer, geometry, materials, and lights, 
//   and should trace a ray through the scene and write the resultant color to a pixel in the image buffer.
////////////////////////////////
///////////////////////////////
// Core raytracer kernel
__global__ void raytraceRay(glm::vec2 resolution, float time, cameraData cam, int maxDepth, glm::vec3* colors,
                            staticGeom* geoms, int numberOfGeoms, material* materials, int numberOfMaterials, 
                            rayState* rayList, int currDepth, int* validRays, int length){
  //need to update for string compaction
  //int x = (blockIdx.x * blockDim.x) + threadIdx.x;
  //int y = (blockIdx.y * blockDim.y) + threadIdx.y;
  //int index = x + (y * resolution.x);
  int index = (blockIdx.x * blockDim.x) + threadIdx.x;
  if(index < length){
  //if((x<=resolution.x && y<=resolution.y)){
    if(rayList[index].isValid == 0){
      return;
    }
    if(currDepth >= maxDepth){//exceeded max depth
       //this contribution is black
      colors[rayList[index].photoIDX] = (colors[rayList[index].photoIDX] * (time - 1.0f)/time) + (glm::vec3(0,0,0) * 1.0f/time);
      rayList[index].isValid = 0;
      validRays[index] = 0;
      return;
    }
    //get variables
    ray thisRay     = rayList[index].RAY;
    glm::vec3 COLOR = rayList[index].color;

    //intersection checks:
    float distToIntersect = FLT_MAX;//infinite distance
    float tmpDist;
    glm::vec3 tmpIntersectPoint, tmpIntersectNormal, intersectPoint, intersectNormal;
    material mat;
    
    for(int i = 0; i < numberOfGeoms; i++){
      if (geoms[i].type == SPHERE){
        tmpDist = sphereIntersectionTest(geoms[i], thisRay, tmpIntersectPoint, tmpIntersectNormal);
      }else if (geoms[i].type == CUBE){
        tmpDist = boxIntersectionTest(   geoms[i], thisRay, tmpIntersectPoint, tmpIntersectNormal);
      }//insert triangles here for meshes
      if (tmpDist != -1 && tmpDist < distToIntersect){ //hit is new closest
        distToIntersect = tmpDist;
        intersectNormal = tmpIntersectNormal;
        intersectPoint  = tmpIntersectPoint;
        mat = materials[geoms[i].materialid];
      }
    }
    //Did I intersect anything?
    if(distToIntersect == FLT_MAX){//miss
      //this contribution is black
      colors[rayList[index].photoIDX] = (colors[rayList[index].photoIDX] * (time - 1.0f)/time) + (glm::vec3(0,0,0) * 1.0f/time);
      rayList[index].isValid = 0;
      validRays[index] = 0;
    }
    //is this a light source?
    if(mat.emittance > 0.001){
      COLOR = COLOR * (mat.color * mat.emittance);
      colors[rayList[index].photoIDX] = (colors[rayList[index].photoIDX] * (time - 1.0f)/time) + (COLOR * 1.0f/time);
      rayList[index].isValid = 0;
      validRays[index] = 0;
      return;
    }
    
    //update variables
    thrust::default_random_engine rng(hash(index * (time + currDepth)));
    thrust::uniform_real_distribution<float> u01(0,1);
    calculateBSDF(thisRay, intersectPoint, intersectNormal, COLOR, mat, (float) u01(rng) ,(float) u01(rng)); 
    //update struct
    rayList[index].RAY   = thisRay;
    rayList[index].color = COLOR;
  }
}

__global__ void compactRays(int* scanRays, rayState* rayList, int* validRays, int length){
  int index = (blockIdx.x * blockDim.x) + threadIdx.x;
  if(index >= length){
    return;
  }
  validRays[index] = 0;
  if(index == 0){//first 
    return;
  }
  rayState newRay = rayList[index];
  __syncthreads();
  if(scanRays[index - 1] < scanRays[index]){
    rayList[scanRays[index]] = newRay;
    validRays[scanRays[index]] = 1;
  }
}


///////////////////////////////////
//////////////////////////////////
// TODO: Finish THIS FUNCTION /// You will have to complete this function to support passing materials and lights to CUDA
////////////////////////////////
///////////////////////////////
// Wrapper for the __global__ call that sets up the kernel calls and does a ton of memory management
void cudaRaytraceCore(uchar4* PBOpos, camera* renderCam, int frame, int iterations, material* materials, int numberOfMaterials, geom* geoms, int numberOfGeoms){
  
  int traceDepth = 10; //determines how many bounces the raytracer traces

  // set up crucial magic
  int tileSize = 8;
  dim3 threadsPerBlock(tileSize, tileSize);
  dim3 fullBlocksPerGrid((int)ceil(float(renderCam->resolution.x)/float(tileSize)), (int)ceil(float(renderCam->resolution.y)/float(tileSize)));
  
  // send image to GPU
  glm::vec3* cudaimage = NULL;
  hipMalloc((void**)&cudaimage,           (int)renderCam->resolution.x * (int)renderCam->resolution.y * sizeof(glm::vec3));
  hipMemcpy( cudaimage, renderCam->image, (int)renderCam->resolution.x * (int)renderCam->resolution.y * sizeof(glm::vec3), hipMemcpyHostToDevice);
  
  // package geometry and materials and sent to GPU
  staticGeom* geomList = new staticGeom[numberOfGeoms];
  for(int i=0; i<numberOfGeoms; i++){
    staticGeom newStaticGeom;
    newStaticGeom.type = geoms[i].type;
    newStaticGeom.materialid = geoms[i].materialid;
    newStaticGeom.translation = geoms[i].translations[frame];
    newStaticGeom.rotation = geoms[i].rotations[frame];
    newStaticGeom.scale = geoms[i].scales[frame];
    newStaticGeom.transform = geoms[i].transforms[frame];
    newStaticGeom.inverseTransform = geoms[i].inverseTransforms[frame];
    geomList[i] = newStaticGeom;
  }
  
  staticGeom* cudageoms = NULL;
  hipMalloc((void**)&cudageoms,   numberOfGeoms * sizeof(staticGeom));
  hipMemcpy( cudageoms, geomList, numberOfGeoms * sizeof(staticGeom), hipMemcpyHostToDevice);
  
  // package camera
  cameraData cam;
  cam.resolution = renderCam->resolution;
  cam.position = renderCam->positions[frame];
  cam.view = renderCam->views[frame];
  cam.up = renderCam->ups[frame];
  cam.fov = renderCam->fov;
  
  // package materials
  material* materialList = NULL;
  hipMalloc((void**) &materialList,   numberOfMaterials * sizeof(material));
  hipMemcpy( materialList, materials, numberOfMaterials * sizeof(material), hipMemcpyHostToDevice);
  
  //allocate Rays
  rayState* rayList = NULL;
  hipMalloc((void**)&rayList, (int)renderCam->resolution.x * (int)renderCam->resolution.y * sizeof(rayState));

  


  // kernel launches
  //Get initial rays
  initializeRay<<<fullBlocksPerGrid, threadsPerBlock>>>(renderCam->resolution, (float)iterations, cam, rayList);
  thrust::device_vector<int> validRays((int)renderCam->resolution.x * (int)renderCam->resolution.y, 1);
  int* thrustArray = thrust::raw_pointer_cast( &validRays[0] );
  int length = thrust::count(validRays.begin(), validRays.end(), 1);//count valid rays
  std::cout << length << "\n";
  thrust::device_vector<int> scanRay((int)renderCam->resolution.x * (int)renderCam->resolution.y, 0);
  int* scanPointer = thrust::raw_pointer_cast( &scanRay[0] );
  int* scanFifty = new int[50];
  
  
  //depth trace with compaction
  for(int i = 0; i <= traceDepth; i++){
    //do one step
    raytraceRay<<<(int)ceil((float)length/64.0f), 64>>>(renderCam->resolution, (float)iterations, cam, traceDepth, cudaimage, cudageoms, numberOfGeoms, materialList, numberOfMaterials, rayList, i, thrustArray, length);
    //build scan
    thrust::exclusive_scan(validRays.begin(), validRays.end(), &scanRay[0]);
    scanPointer = thrust::raw_pointer_cast( &scanRay[0] );
    //compact rays
    compactRays<<<(int)ceil((float)length/64.0f), 64 >>>(scanPointer, rayList, thrustArray, length);
    //update length
    length = thrust::count(validRays.begin(), validRays.end(), 1);//count valid rays
    
    
    hipMemcpy(scanFifty, scanPointer, 50 * sizeof(int), hipMemcpyDeviceToHost);
    
    std::cout <<"scan=" << scanFifty[49] << " " << scanFifty[32] << "\n";
    std::cout  << "\n";
    std::cout << length << "\n";
  }

  //update visual
  sendImageToPBO<<<fullBlocksPerGrid, threadsPerBlock>>>(PBOpos, renderCam->resolution, cudaimage);

  // retrieve image from GPU
  hipMemcpy( renderCam->image, cudaimage, (int)renderCam->resolution.x*(int)renderCam->resolution.y*sizeof(glm::vec3), hipMemcpyDeviceToHost);

  // free up stuff, or else we'll leak memory like a madman
  hipFree( cudaimage );
  hipFree( cudageoms );
  hipFree(materialList); //added
  hipFree(rayList); //added
  free(scanFifty);//added
  delete geomList;

  // make certain the kernel has completed
  hipDeviceSynchronize();

  checkCUDAError("Kernel failed!");
}
